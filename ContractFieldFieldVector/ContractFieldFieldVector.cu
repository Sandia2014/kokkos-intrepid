#include "hip/hip_runtime.h"
// -*- C++ -*-
// ArrayOfDotProducts.cc
// a huge comparison of different ways of doing an array of dot products
// Jeff Amelang, 2014

// c junk
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <cstdio>
#include <unistd.h>

// c++ junk
#include <vector>
#include <algorithm>
#include <chrono>
#include <random>
#include <string>
#include <array>
#include <fstream>
using std::string;
using std::vector;
using std::array;

// header file for openmp
#include <omp.h>

#ifdef ENABLE_KOKKOS
#include <Kokkos_Core.hpp>
#endif // ENABLE_KOKKOS

enum CudaStyle {CudaStyle_Independent,
                CudaStyle_Reduction};

enum ClearCacheStyle {ClearCacheAfterEveryRepeat,
                      DontClearCacheAfterEveryRepeat};

string
convertCudaStyleToString(const CudaStyle cudaStyle) {
  switch (cudaStyle) {
  case CudaStyle_Independent:
    return string("CudaStyle_Independent");
  case CudaStyle_Reduction:
    return string("CudaStyle_Reduction");
  default:
    fprintf(stderr, "invalid cuda style\n");
    exit(1);
  };
}

// stolen from http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline
void
gpuAssert(const hipError_t code, const char *file, const int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPU Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort == true) {
      exit(code);
    }
  }
}

timespec
getTimePoint() {
  timespec timepoint;
  clock_gettime(CLOCK_MONOTONIC, &timepoint);
  return timepoint;
}

// yay for having to use pre-c++11 timing because of nvcc
double
getElapsedTime(const timespec & start, const timespec & end) {
  timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}





__global__
void
doCudaClearCache_kernel(const unsigned int junkDataSize,
                        const int * const __restrict__ dev_junkDataToClearTheCache,
                        int * dev_result) {
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  int partialSum = 0;
  while (index < junkDataSize) {
    partialSum += dev_junkDataToClearTheCache[index];
    index += blockDim.x * gridDim.x;
  }
  atomicAdd(dev_result, partialSum);
}

__global__
void
doCudaDotProducts_Independent_kernel(const unsigned int numberOfContractions,
                                     const unsigned int maxNumberOfContractions,
                                     const unsigned int contractionSize,
                                     const float * const __restrict__ dev_contractionData_LayoutLeft_A,
                                     const float * const __restrict__ dev_contractionData_LayoutLeft_B,
                                     float * dev_contractionResults,
                                     const unsigned int l,
                                     const unsigned int r,
                                     const unsigned int q,
                                     const unsigned int i) {
  unsigned int contractionIndex = blockIdx.x * blockDim.x + threadIdx.x;
  while (contractionIndex < numberOfContractions) {
    int cellNum = contractionIndex / (l*r);
    int fieldsIndex = contractionIndex % (l*r);
    int leftFieldNum = fieldsIndex / r;
    int rightFieldNum = fieldsIndex % r;

    double tmpVal = 0;
    for (int qp = 0; qp < q; qp++) {
      for (int iVec = 0; iVec < i; iVec++) {
        tmpVal += dev_contractionData_LayoutLeft_A[cellNum * l * q * i +
                                        leftFieldNum * q * i + qp * i + iVec]
                  *dev_contractionData_LayoutLeft_B[cellNum * r * q * i +
                                    rightFieldNum * q * i + qp * i + iVec];
      } //D-loop
    } // P-loop

    dev_contractionResults[cellNum * l * r + leftFieldNum * r + rightFieldNum] = tmpVal;
  }
}


__global__
void
doCudaDotProducts_Reduction_kernel(const unsigned int numberOfDotProducts,
                                   const unsigned int dotProductSize,
                                   const float * const __restrict__ dev_dotProductData_LayoutRight_A,
                                   const float * const __restrict__ dev_dotProductData_LayoutRight_B,
                                   float * dev_dotProductResults) {

  extern __shared__ float sharedMemory[];

  unsigned int dotProductIndex = blockIdx.x;
  while (dotProductIndex < numberOfDotProducts) {

    // goal: compute the contribution to the dot product from this thread
    const unsigned int shortcutIndex = dotProductIndex * dotProductSize;
    float partialSum = 0;
    unsigned int entryIndex = threadIdx.x;
    while (entryIndex < dotProductSize) {
      const unsigned int index = shortcutIndex + entryIndex;
      partialSum +=
        dev_dotProductData_LayoutRight_A[index] *
        dev_dotProductData_LayoutRight_B[index];
      entryIndex += blockDim.x;
    }
    // set this thread's value
    sharedMemory[threadIdx.x] = partialSum;

    // goal: reduce the warp's contribution to one number and add it to the
    //  dot product.

    // each warp does its own reduction
    const unsigned int warpIndex = threadIdx.x / 32;
    const unsigned int indexWithinWarp = threadIdx.x - warpIndex * 32;
    const unsigned int warpStartingIndexInSharedMemory = warpIndex * 32;
    // the first quarter of the threads in the warp make small partial sums
    if (indexWithinWarp < 8) {
      const int i = indexWithinWarp * 4;
      partialSum  = sharedMemory[warpStartingIndexInSharedMemory + i];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + i + 1];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + i + 2];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + i + 3];
      sharedMemory[warpStartingIndexInSharedMemory + i] = partialSum;
    }
    // the first thread in the warp reduces the 8 partial sums
    if (indexWithinWarp == 0) {
      partialSum += sharedMemory[warpStartingIndexInSharedMemory +  4];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory +  8];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 12];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 16];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 20];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 24];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 28];
      // and adds it to the global sum
      atomicAdd(&dev_dotProductResults[dotProductIndex], partialSum);
    }

    // move on to the next dot product
    dotProductIndex += gridDim.x;
  }
}

void
writeTimesMatrixToFile(const vector<vector<float> > & times,
                       const string filename) {

  const unsigned int numberOfDotProductSizes = times.size();
  // yeah, yeah, kinda unsafe
  const unsigned int numberOfMemorySizes = times[0].size();
  char sprintfBuffer[500];
  sprintf(sprintfBuffer, "%s.csv", filename.c_str());
  FILE* file = fopen(sprintfBuffer, "w");
  for (unsigned int dotProductSizeIndex = 0;
       dotProductSizeIndex < numberOfDotProductSizes;
       ++dotProductSizeIndex) {
    for (unsigned int memorySizeIndex = 0;
         memorySizeIndex < numberOfMemorySizes;
         ++memorySizeIndex) {
      if (memorySizeIndex > 0) {
        fprintf(file, ", ");
      }
      fprintf(file, "%10.4e", times[dotProductSizeIndex][memorySizeIndex]);
    }
    fprintf(file, "\n");
  }
  fclose(file);
}

void
checkAnswer(const vector<float> & correctResults,
            const vector<float> & dotProductResults,
            const unsigned int dotProductSize,
            const unsigned int memorySize,
            const string flavorName) {
  for (unsigned int dotProductIndex = 0;
       dotProductIndex < correctResults.size();
       ++dotProductIndex) {
    if (std::abs(correctResults[dotProductIndex] -
                 dotProductResults[dotProductIndex]) /
        std::abs(correctResults[dotProductIndex]) > 1e-4) {
      fprintf(stderr, "invalid answer for dot product index %u for "
              "flavor %s, "
              "should be %e but we have %e, "
              "dotProductSize = %u, memorySize = %8.2e\n",
              dotProductIndex, flavorName.c_str(),
              correctResults[dotProductIndex],
              dotProductResults[dotProductIndex],
              dotProductSize, float(memorySize));
      exit(1);
    }
  }
}

double
runCudaTest(const CudaStyle cudaStyle,
            const unsigned int numberOfThreadsPerBlock,
            const unsigned int numberOfRepeats,
            const unsigned int maxNumberOfCudaBlocks,
            const unsigned int numberOfContractions,
            const unsigned int maxNumberOfContractions,
            const unsigned int contractionSize,
            const unsigned int memorySize,
            const vector<float> & correctResults,
            const ClearCacheStyle clearCacheStyle,
            const int * const dev_junkDataToClearTheCache,
            const unsigned int junkDataSize,
            const float * const dev_contractionData_A,
            const float * const dev_contractionData_B,
            int * const dev_junkDataCounter,
            unsigned int * const totalNumberOfRepeats,
            float * const dev_dotProductResults,
            vector<float> * const contractionResults,
            const unsigned int l,
            const unsigned int r,
            const unsigned int q,
            const unsigned int i) {
  const unsigned int numberOfBlocks =
    min(maxNumberOfCudaBlocks,
        (unsigned int)ceil(numberContractions/float(numberOfThreadsPerBlock)));

  timespec tic;
  double totalElapsedTime = 0;
  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
    *totalNumberOfRepeats = *totalNumberOfRepeats + 1;
    if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
         repeatIndex == 1) ||
        clearCacheStyle == ClearCacheAfterEveryRepeat) {
      tic = getTimePoint();
    }

    // do the actual calculation
    if (cudaStyle == CudaStyle_Independent) {
      doCudaContractions_Independent_kernel<<<numberOfBlocks,
        numberOfThreadsPerBlock>>>(numberOfContractions,
                                   maxNumberOfContractions,
                                   contractionSize,
                                   dev_contractionData_A,
                                   dev_contractionData_B,
                                   dev_contractionResults,
                                   l,
                                   r,
                                   q,
                                   i);
    } else if (cudaStyle == CudaStyle_Reduction) {
      doCudaContractions_Reduction_kernel<<<numberOfBlocks,
        numberOfThreadsPerBlock,
        numberOfThreadsPerBlock * sizeof(float)>>>(numberOfContractions,
                                                   contractionSize,
                                                   dev_contractionData_A,
                                                   dev_contractionData_B,
                                                   dev_contractionResults);
    } else {
      fprintf(stderr, "unknown cuda style\n");
      exit(1);
    }

    // wait for the kernel launch
    checkCudaError(hipPeekAtLastError());
    checkCudaError(hipDeviceSynchronize());
    if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
      const timespec toc = getTimePoint();
      const float elapsedTime = getElapsedTime(tic, toc);
      totalElapsedTime += elapsedTime;

      const unsigned int junkNumberOfBlocks =
        min(maxNumberOfCudaBlocks,
            (unsigned int)ceil(junkDataSize/float(numberOfThreadsPerBlock)));
      doCudaClearCache_kernel<<<junkNumberOfBlocks,
        numberOfThreadsPerBlock>>>(junkDataSize,
                                   dev_junkDataToClearTheCache,
                                   dev_junkDataCounter);
      // wait for the kernel launch
      checkCudaError(hipPeekAtLastError());
      checkCudaError(hipDeviceSynchronize());
    }
  }
  if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
    const timespec toc = getTimePoint();
    const float elapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
    totalElapsedTime = elapsedTime;
  }
  // copy over the results from the gpu to the cpu
  checkCudaError(hipMemcpy(&dotProductResults->at(0), dev_dotProductResults,
                            numberOfDotProducts * sizeof(float),
                            hipMemcpyDeviceToHost));
  // check the results
  checkAnswer(correctResults, *dotProductResults,
              dotProductSize, memorySize,
              convertCudaStyleToString(cudaStyle));

  // scrub the results
  std::fill(dotProductResults->begin(),
            dotProductResults->end(),
            std::numeric_limits<float>::quiet_NaN());
  checkCudaError(hipMemcpy(dev_dotProductResults, &dotProductResults->at(0),
                            numberOfDotProducts * sizeof(float),
                            hipMemcpyHostToDevice));

  return totalElapsedTime;
}

double
runSwitchingCudaTest(const unsigned int numberOfRepeats,
                     const unsigned int maxNumberOfCudaBlocks,
                     const unsigned int numberOfDotProducts,
                     const unsigned int maxNumberOfDotProducts,
                     const unsigned int dotProductSize,
                     const unsigned int memorySize,
                     const vector<float> & correctResults,
                     const ClearCacheStyle clearCacheStyle,
                     const int * const dev_junkDataToClearTheCache,
                     const unsigned int junkDataSize,
                     const float * const dev_dotProductData_LayoutLeft_A,
                     const float * const dev_dotProductData_LayoutLeft_B,
                     const float * const dev_dotProductData_LayoutRight_A,
                     const float * const dev_dotProductData_LayoutRight_B,
                     int * const dev_junkDataCounter,
                     unsigned int * const totalNumberOfRepeats,
                     float * const dev_dotProductResults,
                     vector<float> * const dotProductResults) {
  // if i can't saturate occupancy, do the reduction version
  // i got this number by just looking at where the plots crossed, where
  //  the reduction style actually starts beating the independent.
  if (numberOfDotProducts < 200) {
    const unsigned int numberOfThreadsPerBlock =
      std::min(unsigned(1024),
               unsigned(ceil(dotProductSize / 32.)) * 32);
    return
      runCudaTest(CudaStyle_Reduction,
                  numberOfThreadsPerBlock,
                  numberOfRepeats,
                  maxNumberOfCudaBlocks,
                  numberOfDotProducts,
                  maxNumberOfDotProducts,
                  dotProductSize,
                  memorySize,
                  correctResults,
                  clearCacheStyle,
                  dev_junkDataToClearTheCache,
                  junkDataSize,
                  dev_dotProductData_LayoutRight_A,
                  dev_dotProductData_LayoutRight_B,
                  dev_junkDataCounter,
                  totalNumberOfRepeats,
                  dev_dotProductResults,
                  dotProductResults);
  } else {
    const unsigned int numberOfThreadsPerBlock = 1024;
    return
      runCudaTest(CudaStyle_Independent,
                  numberOfThreadsPerBlock,
                  numberOfRepeats,
                  maxNumberOfCudaBlocks,
                  numberOfDotProducts,
                  maxNumberOfDotProducts,
                  dotProductSize,
                  memorySize,
                  correctResults,
                  clearCacheStyle,
                  dev_junkDataToClearTheCache,
                  junkDataSize,
                  dev_dotProductData_LayoutLeft_A,
                  dev_dotProductData_LayoutLeft_B,
                  dev_junkDataCounter,
                  totalNumberOfRepeats,
                  dev_dotProductResults,
                  dotProductResults);
  }
}





#ifdef ENABLE_KOKKOS

template <class DeviceType, class KokkosJunkVector>
struct KokkosFunctor_ClearCache {

  typedef size_t     value_type;
  typedef DeviceType device_type;

  KokkosJunkVector _junkDataToClearTheCache;

  KokkosFunctor_ClearCache(KokkosJunkVector dev_junkDataToClearTheCache) :
    _junkDataToClearTheCache(dev_junkDataToClearTheCache) {
  }

  KOKKOS_INLINE_FUNCTION
  void operator()(const unsigned int index,
                  value_type & junkDataCounter) const {
    junkDataCounter += _junkDataToClearTheCache(index);
  }

private:
  KokkosFunctor_ClearCache();

};

template<class DeviceType, class LeftViewType, class RightViewType, class OutputViewType>
struct KokkosFunctor_Independent {

  typedef DeviceType device_type;
  LeftViewType _leftInput;
  RightViewType _rightInput;
  OutputViewType _output;
  int _numCells;
  int _numPoints;
  int _numLeftFields;
  int _numRightFields;
  int _dimVec;

  KokkosFunctor_Independent(LeftViewType leftInput,
  RightViewType rightInput,
  OutputViewType output,
  int c,
  int l,
  int r,
  int q,
  int i) :
  _leftInput(leftInput),
  _rightInput(rightInput),
  _output(output),
  _numCells(c),
  _numPoints(q),
  _numLeftFields(l),
  _numRightFields(r),
  _dimVec(i)
  {
    // Nothing to do
  }

  // Parallelize over c-loop
  KOKKOS_INLINE_FUNCTION
  void operator()(const unsigned int elementIndex) const {
    int cellNum = elementIndex / (_numLeftFields*_numRightFields);
    int fieldsIndex = elementIndex % (_numLeftFields*_numRightFields);
    int leftFieldNum = fieldsIndex / _numRightFields;
    int rightFieldNum = fieldsIndex % _numRightFields;

    double tmpVal = 0;
    for (int qp = 0; qp < _numPoints; qp++) {
      for (int iVec = 0; iVec < _dimVec; iVec++) {
        tmpVal += _leftInput(cellNum, leftFieldNum, qp, iVec)*_rightInput(cellNum,rightFieldNum,qp, iVec);
      } //D-loop
    } // P-loop

    _output(cellNum, leftFieldNum, rightFieldNum) = tmpVal;
  }
};

template <class DeviceType, class KokkosDotProductData_Left, class KokkosDotProductData_Right>
double
runKokkosTest(const unsigned int cellSize,
	      const unsigned int numberOfRepeats,
              const unsigned int memorySize,
              const unsigned int numCells,
              const unsigned int l,
              const unsigned int r,
              const unsigned int q,
              const unsigned int i,
              const vector<float> & dotProductData_LayoutRight_A,
              const vector<float> & dotProductData_LayoutRight_B,
              const vector<float> & correctResults,
              const string & kokkosFlavor,
              const ClearCacheStyle clearCacheStyle,
              const vector<int> & junkDataToClearTheCache,
              size_t * junkDataCounter,
              unsigned int * const totalNumberOfRepeats,
              vector<float> * dotProductResults) {

  const unsigned int junkDataSize = junkDataToClearTheCache.size();

  typedef typename KokkosDotProductData_Left::HostMirror     KokkosDotProductData_Host_Left;
  typedef typename KokkosDotProductData_Right::HostMirror     KokkosDotProductData_Host_Right;

  typedef Kokkos::View<float***, DeviceType>              KokkosDotProductResults;
  typedef typename KokkosDotProductResults::HostMirror  KokkosDotProductResults_Host;
  typedef Kokkos::View<int*, DeviceType>                KokkosJunkVector;
  typedef typename KokkosJunkVector::HostMirror         KokkosJunkVector_Host;

  KokkosDotProductData_Left dev_kokkosDotProductData_A("kokkos data A",
                                                  numCells,
                                                  l, q, i);
  KokkosDotProductData_Host_Left kokkosDotProductData_A =
    Kokkos::create_mirror_view(dev_kokkosDotProductData_A);

  KokkosDotProductData_Right dev_kokkosDotProductData_B("kokkos data B",
                                                  numCells,
                                                  r,
                                                  q,
                                                  i);
  KokkosDotProductData_Host_Right kokkosDotProductData_B =
    Kokkos::create_mirror_view(dev_kokkosDotProductData_B);

  KokkosDotProductResults dev_kokkosDotProductResults("kokkos dot product results",
                                                      numCells,l, r);
  KokkosDotProductResults_Host kokkosDotProductResults =
    Kokkos::create_mirror_view(dev_kokkosDotProductResults);

  KokkosJunkVector dev_kokkosJunkDataToClearTheCache("kokkos junk data to clear cache",
                                                     junkDataSize);
  KokkosJunkVector_Host kokkosJunkDataToClearTheCache =
    Kokkos::create_mirror_view(dev_kokkosJunkDataToClearTheCache);

  // copy the data into the device views and ship them over
  for (unsigned int dotProductIndex = 0;
       dotProductIndex < numCells; ++dotProductIndex) {
    for (unsigned int lbf = 0; lbf < l; ++lbf) {
      for(unsigned int rbf = 0; rbf < r; ++rbf) {
        for(unsigned int qp = 0; qp < q; ++qp) {
          for(unsigned int iVec = 0; iVec < i; ++iVec) {
            kokkosDotProductData_A(dotProductIndex, lbf, qp, iVec) =
            dotProductData_LayoutRight_A[dotProductIndex * l * q * i +
            lbf * q * i + qp * i + iVec];
            kokkosDotProductData_B(dotProductIndex, rbf, qp, iVec) =
            dotProductData_LayoutRight_B[dotProductIndex * r * q * i +
            rbf * q * i + qp * i + iVec];
          }
        }
      }
    }
  }
  Kokkos::deep_copy(dev_kokkosDotProductData_A, kokkosDotProductData_A);
  Kokkos::deep_copy(dev_kokkosDotProductData_B, kokkosDotProductData_B);

  // copy the data into the device views and ship them over
  for (unsigned int junkDataIndex = 0;
       junkDataIndex < junkDataSize; ++junkDataIndex) {
    kokkosJunkDataToClearTheCache(junkDataIndex) =
      junkDataToClearTheCache[junkDataIndex];
  }
  Kokkos::deep_copy(dev_kokkosJunkDataToClearTheCache, kokkosJunkDataToClearTheCache);

  KokkosFunctor_ClearCache<DeviceType,
                           KokkosJunkVector>
    kokkosFunctor_ClearCache(dev_kokkosJunkDataToClearTheCache);

    // breaking formatting convention because holy freak that's long
    KokkosFunctor_Independent<DeviceType, KokkosDotProductData_Left,
                            KokkosDotProductData_Right, KokkosDotProductResults>
    kokkosFunctor_Independent(dev_kokkosDotProductData_A,
    dev_kokkosDotProductData_B,
    dev_kokkosDotProductResults,
    numCells, l, r,q, i);

  timespec tic;
  double totalElapsedTime = 0;
  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
    *totalNumberOfRepeats = *totalNumberOfRepeats + 1;
    if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
         repeatIndex == 1) ||
        clearCacheStyle == ClearCacheAfterEveryRepeat) {
      tic = getTimePoint();
    }

    // actually do the calculation
    Kokkos::parallel_for(numCells*r*l, kokkosFunctor_Independent);

    // wait for this repeat's results to finish
    Kokkos::fence();

    if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
      const timespec toc = getTimePoint();
      const float elapsedTime = getElapsedTime(tic, toc);
      totalElapsedTime += elapsedTime;

      // attempt to scrub all levels of cache
      size_t partialJunkDataCounter = 0;
      Kokkos::parallel_reduce(junkDataSize, kokkosFunctor_ClearCache,
                              partialJunkDataCounter);
      *junkDataCounter += partialJunkDataCounter;
    }
  }
  if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
    const timespec toc = getTimePoint();
    totalElapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
  }
  // copy over the results from the device to the host
  Kokkos::deep_copy(kokkosDotProductResults, dev_kokkosDotProductResults);
  for (unsigned int dotProductIndex = 0;
       dotProductIndex < numCells; ++dotProductIndex) {
         for(unsigned int lbf = 0; lbf < l; ++lbf) {
           for(unsigned int rbf = 0; rbf < r; ++rbf) {
             dotProductResults->at(dotProductIndex*l*r+ lbf*r + rbf) =
             kokkosDotProductResults(dotProductIndex, lbf, rbf);
        }
    }
  }
  // check the results
  checkAnswer(correctResults, *dotProductResults,
              cellSize, memorySize,
              kokkosFlavor);

  // scrub the results
  std::fill(dotProductResults->begin(),
            dotProductResults->end(),
            std::numeric_limits<float>::quiet_NaN());

  return totalElapsedTime;
}

#endif // ENABLE_KOKKOS



int main(int argc, char* argv[]) {

#ifdef ENABLE_KOKKOS
  Kokkos::initialize(argc, argv);
#endif

  // ===============================================================
  // ********************** < input> ******************************
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
  const vector<unsigned int> contractionSizes =
    {{16, 25, 49, 144, 225, 361, 529, 729, 961}};
  const array<float, 2> memorySizeExtrema = {{1e6, 1e9}};
  const unsigned int numberOfMemorySizes = 20;
  const unsigned int maxNumberOfCudaBlocks = unsigned(1e4);
  const unsigned int l = 8;
  const unsigned int r = 8;

  unsigned int cellSize;
  const ClearCacheStyle clearCacheStyle =
    ClearCacheAfterEveryRepeat;
  const unsigned int numberOfRepeats =
    (clearCacheStyle == ClearCacheAfterEveryRepeat) ? 10 : 250;
  const string machineName = "shadowfax";
  const string prefix = "data/ArrayOfContractions_";
  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  // ********************** </input> ******************************
  // ===============================================================

  // derive some values from the inputs
  const unsigned int numberOfContractionSizes = contractionSizes.size();
  const string clearCacheStyleString =
    (clearCacheStyle == ClearCacheAfterEveryRepeat) ? "clearCache" :
    "dontClearCache";
  const string suffix = "_" + clearCacheStyleString + "_" + machineName;

  // create the actual sizes
  vector<vector <unsigned int>> memorySizes(numberOfMemorySizes,
                                          vector<unsigned int>(numberOfContractionSizes, 0));
  for (unsigned int memorySizeIndex = 0;
       memorySizeIndex < numberOfMemorySizes; ++memorySizeIndex) {
    const float percent = memorySizeIndex / float(numberOfMemorySizes - 1);
    const float minLog = log10(memorySizeExtrema[0]);
    const float maxLog = log10(memorySizeExtrema[1]);
    const float thisLog = minLog + percent * (maxLog - minLog);
    const unsigned int maxContractionSize = contractionSizes.back();
    // memory size is linear on a log scale, but rounded to a multiple of the
    //  largest dot product size
    const unsigned int desiredMemorySizeInBytes = pow(10., thisLog);
    // now, in this amount of memory i have to fit two vectors of data
    // that are multiples of the max dot product size
    for(unsigned int contractionIndex = 0; contractionIndex < numberOfContractionSizes;
        ++contractionIndex){

          cellSize = 2*l*(contractionSizes[contractionIndex])*sizeof(float)
                    + 2*r*(contractionSizes[contractionIndex])*sizeof(float);
          const unsigned int memorySizeInBytes =
          unsigned(desiredMemorySizeInBytes /
               cellSize) * cellSize;
          (memorySizes[memorySizeIndex]).at(contractionIndex) = memorySizeInBytes;
   }
  }

  // create a c++11 random number generator
  std::mt19937 randomNumberEngine;
  std::uniform_real_distribution<float> randomNumberGenerator(0, 1);

  // these are just containers for storing the numbers we'll be plotting.
  // i feel a little dirty using a vector<vector>, but i don't want to introduce
  //  a dependence on eigen or something for a real matrix.
  vector<vector<float> >
    contractionSizeMatrix(numberOfContractionSizes,
                         vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    numberOfContractionsMatrix(numberOfContractionSizes,
                              vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    memorySizeMatrix(numberOfContractionSizes,
                     vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    serialTimesMatrix(numberOfContractionSizes,
                      vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    ompTimesMatrix(numberOfContractionSizes,
                   vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    cudaIndependent_TimesMatrix(numberOfContractionSizes,
                                vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    cudaReduction_TimesMatrix(numberOfContractionSizes,
                              vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    cudaSwitchingTimesMatrix(numberOfContractionSizes,
                             vector<float>(numberOfMemorySizes, 0));
#ifdef ENABLE_KOKKOS
  vector<vector<float> >
    kokkosOmpTimesMatrix(numberOfContractionSizes,
                         vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    kokkosCudaIndependentTimesMatrix(numberOfContractionSizes,
                                     vector<float>(numberOfMemorySizes, 0));
#endif

  // create some junk data to use in clearing the cache
  size_t junkDataCounter = 0;
  const size_t junkDataSize = 1e7;
  vector<int> junkDataToClearTheCache(junkDataSize, 0);
  for (unsigned int i = 0; i < junkDataSize/100; ++i) {
    junkDataToClearTheCache[(rand() / float(RAND_MAX))*junkDataSize] = 1;
  }
  int * dev_junkDataToClearTheCache;
  checkCudaError(hipMalloc((void **) &dev_junkDataToClearTheCache,
                            junkDataSize * sizeof(int)));
  checkCudaError(hipMemcpy(dev_junkDataToClearTheCache,
                            &junkDataToClearTheCache[0],
                            junkDataSize * sizeof(int),
                            hipMemcpyHostToDevice));
  int * dev_junkDataCounter;
  checkCudaError(hipMalloc((void **) &dev_junkDataCounter,
                            sizeof(int)));
  {
    int temp = 0;
    checkCudaError(hipMemcpy(dev_junkDataCounter,
                              &temp,
                              sizeof(int),
                              hipMemcpyHostToDevice));
  }

  unsigned int totalNumberOfRepeats = 0;

  // for each dot product size
  for (unsigned int contractionSizeIndex = 0;
       contractionSizeIndex < numberOfContractionSizes;
       ++contractionSizeIndex) {
    const unsigned int contractionSize = contractionSizes[contractionSizeIndex];

    const unsigned int q = pow(contractionSize, 1/2);
    const unsigned int i = q;
    const timespec thisSizesTic = getTimePoint();

    // allocate and initialize the largest amount of memory we'll need, then on
    //  each size we'll just use subsets of this memory.
    const unsigned int maxNumberOfContractions =
      memorySizes.back().at(contractionSizeIndex) / 4 / sizeof(float) / contractionSize;
    vector<float> contractionData_LayoutRight_A(maxNumberOfContractions * l * contractionSize);
    vector<float> contractionData_LayoutRight_B(maxNumberOfContractions * r * contractionSize);
    vector<float> contractionData_LayoutLeft_A(contractionData_LayoutRight_A.size());
    vector<float> contractionData_LayoutLeft_B(contractionData_LayoutRight_B.size());

    for (unsigned int contractionIndex = 0;
         contractionIndex < maxNumberOfContractions; ++contractionIndex) {
      for (unsigned int entryIndex = 0;
           entryIndex < contractionSize; ++entryIndex) {

        for (unsigned int rbf = 0; rbf < r; ++rbf) {

        const unsigned int layoutRightIndex_A =
          contractionIndex * contractionSize * r + rbf * contractionSize + entryIndex;

        const unsigned int layoutLeftIndex_A =
          entryIndex * contractionSize * r + rbf * contractionSize + contractionIndex;

        contractionData_LayoutRight_A[layoutRightIndex_A] =
          randomNumberGenerator(randomNumberEngine);

        contractionData_LayoutLeft_A[layoutLeftIndex_A] =
          randomNumberGenerator(randomNumberEngine);
        }

        for(unsigned int lbf = 0; lbf < l; ++lbf) {

          const unsigned int layoutRightIndex_B =
          contractionIndex * contractionSize * l + lbf * contractionSize + entryIndex;

          const unsigned int layoutLeftIndex_B =
          entryIndex * contractionSize * l + lbf * contractionSize + contractionIndex;

          contractionData_LayoutRight_B[layoutRightIndex_B] =
            randomNumberGenerator(randomNumberEngine);

          contractionData_LayoutLeft_B[layoutLeftIndex_B] =
            randomNumberGenerator(randomNumberEngine);
        }
      }
    }
    vector<float> contractionResults(maxNumberOfContractions*l*r,
                                    std::numeric_limits<float>::quiet_NaN());


    // now, because we'll be working with cuda stuff, also allocate the inputs
    //  and output on the gpu and copy them over
    float * dev_contractionData_LayoutRight_A;
    checkCudaError(hipMalloc((void **) &dev_contractionData_LayoutRight_A,
                              maxNumberOfContractions * contractionSize * sizeof(float)));
    checkCudaError(hipMemcpy(dev_contractionData_LayoutRight_A,
                              &contractionData_LayoutRight_A[0],
                              maxNumberOfContractions * contractionSize * sizeof(float),
                              hipMemcpyHostToDevice));
    float * dev_contractionData_LayoutRight_B;
    checkCudaError(hipMalloc((void **) &dev_contractionData_LayoutRight_B,
                              maxNumberOfContractions * contractionSize * sizeof(float)));
    checkCudaError(hipMemcpy(dev_contractionData_LayoutRight_B,
                              &contractionData_LayoutRight_B[0],
                              maxNumberOfContractions * contractionSize * sizeof(float),
                              hipMemcpyHostToDevice));
    float * dev_contractionResults;
    checkCudaError(hipMalloc((void **) &dev_contractionResults,
                              maxNumberOfContractions * sizeof(float)));
    checkCudaError(hipMemcpy(dev_contractionResults, &contractionResults[0],
                              maxNumberOfContractions * sizeof(float),
                              hipMemcpyHostToDevice));
    // make and populate the LayoutLeft versions
    float * dev_contractionData_LayoutLeft_A;
    checkCudaError(hipMalloc((void **) &dev_contractionData_LayoutLeft_A,
                              maxNumberOfContractions * contractionSize * sizeof(float)));
    checkCudaError(hipMemcpy(dev_contractionData_LayoutLeft_A,
                              &contractionData_LayoutLeft_A[0],
                              maxNumberOfContractions * contractionSize * sizeof(float),
                              hipMemcpyHostToDevice));
    float * dev_contractionData_LayoutLeft_B;
    checkCudaError(hipMalloc((void **) &dev_contractionData_LayoutLeft_B,
                              maxNumberOfContractions * contractionSize * sizeof(float)));
    checkCudaError(hipMemcpy(dev_contractionData_LayoutLeft_B,
                              &contractionData_LayoutLeft_B[0],
                              maxNumberOfContractions * contractionSize * sizeof(float),
                              hipMemcpyHostToDevice));

    // for each memory size
    for (unsigned int memorySizeIndex = 0;
         memorySizeIndex < numberOfMemorySizes;
         ++memorySizeIndex) {
      const unsigned int memorySize = memorySizes[memorySizeIndex].at(contractionSizeIndex);
      const unsigned int numCells =
       memorySize / (2*sizeof(float)*r*contractionSize+2*sizeof(float)*l*contractionSize);
      if (memorySize != numCells * (2*sizeof(float)*r*contractionSize+2*sizeof(float)*l*contractionSize)) {
        fprintf(stderr, "invalid memory size of %u for dot product size of "
                "%u because it doesn't divide evenly, remainder is %zu\n",
                memorySize, contractionSize,
                memorySize % (2*sizeof(float)*r*contractionSize+2*sizeof(float)*l*contractionSize));
        exit(1);
      }

      // ===============================================================
      // ********************** < do serial> ***************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        timespec tic;
        for (unsigned int repeatIndex = 0;
             repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
          ++totalNumberOfRepeats;
          if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
               repeatIndex == 1) ||
              clearCacheStyle == ClearCacheAfterEveryRepeat) {
            tic = getTimePoint();
          }

          for (int cl = 0; cl < numCells; cl++) {
            for (int lbf = 0; lbf < l; lbf++) {
              for (int rbf = 0; rbf < r; rbf++) {

                double tmpVal = 0;
                for (int qp = 0; qp < q; qp++) {
                  for (int iVec = 0; iVec < i; iVec++) {
                    tmpVal += contractionData_LayoutRight_A[cl*l*q*i+lbf*q*i+qp*i+iVec]
		    		*contractionData_LayoutRight_B[cl*r*q*i+rbf*q*i+qp*i+iVec];
                  } //D-loop
                } // P-loop
                contractionResults[cl*l*r+lbf*r+rbf] = tmpVal;
              } // R-loop
            } // L-loop
          }

          if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
            const timespec toc = getTimePoint();
            const float elapsedTime = getElapsedTime(tic, toc);
            serialTimesMatrix[contractionSizeIndex][memorySizeIndex] += elapsedTime;

            junkDataCounter +=
              std::accumulate(junkDataToClearTheCache.begin(),
                              junkDataToClearTheCache.end(), size_t(0));
          }
        }
        if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
          const timespec toc = getTimePoint();
          const float elapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
          serialTimesMatrix[contractionSizeIndex][memorySizeIndex] = elapsedTime;
        }
      }
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ********************** </do serial> ***************************
      // ===============================================================

      const vector<float> correctResults = contractionResults;
      // scrub the results
      std::fill(contractionResults.begin(),
                contractionResults.end(),
                std::numeric_limits<float>::quiet_NaN());

      // ===============================================================
      // ********************** < do omp> ******************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
     /*  {

        timespec tic;
        for (unsigned int repeatIndex = 0;
             repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
          ++totalNumberOfRepeats;
          if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
               repeatIndex == 1) ||
              clearCacheStyle == ClearCacheAfterEveryRepeat) {
            tic = getTimePoint();
          }

          // do the actual calculation
#pragma omp parallel for default(none)                                  \
  shared(dotProductData_LayoutRight_A, dotProductData_LayoutRight_B,    \
         dotProductResults)
          for (unsigned int dotProductIndex = 0;
               dotProductIndex < numberOfDotProducts;
               ++dotProductIndex) {
            const unsigned int shortcutIndex = dotProductIndex * dotProductSize;
            float sum = 0;
            for (unsigned int entryIndex = 0;
                 entryIndex < dotProductSize; ++entryIndex) {
              sum +=
                dotProductData_LayoutRight_A[shortcutIndex + entryIndex] *
                dotProductData_LayoutRight_B[shortcutIndex + entryIndex];
            }
            dotProductResults[dotProductIndex] = sum;
          }

          if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
            const timespec toc = getTimePoint();
            const float elapsedTime = getElapsedTime(tic, toc);
            ompTimesMatrix[dotProductSizeIndex][memorySizeIndex] += elapsedTime;

            // attempt to scrub all levels of cache
#pragma omp parallel default(none)                      \
  shared(junkDataCounter, junkDataToClearTheCache)
            {
              const size_t thisThreadsJunkDataCounter =
                std::accumulate(junkDataToClearTheCache.begin(),
                                junkDataToClearTheCache.end(), size_t(0));
              // only one thread adds the junk counter so that the total
              //  at the end is not a function of the number of threads.
#pragma omp single
              junkDataCounter += thisThreadsJunkDataCounter;
            }
          }
        }
        if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
          const timespec toc = getTimePoint();
          const float elapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
          ompTimesMatrix[dotProductSizeIndex][memorySizeIndex] = elapsedTime;
          // check the results
          checkAnswer(correctResults, dotProductResults,
                      dotProductSize, memorySize,
                      string("omp"));
        }
      }
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ********************** </do omp> ******************************
      // ===============================================================

      // scrub the results
      std::fill(dotProductResults.begin(),
                dotProductResults.end(),
                std::numeric_limits<float>::quiet_NaN());
      checkCudaError(hipMemcpy(dev_dotProductResults, &dotProductResults[0],
                                maxNumberOfDotProducts * sizeof(float),
                                hipMemcpyHostToDevice));
                                */
      // ===============================================================
      // ***************** < do cuda independent> **********************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        const unsigned int numberOfThreadsPerBlock = 1024;

        cudaIndependent_TimesMatrix[contractionSizeIndex][memorySizeIndex] =
          runCudaTest(CudaStyle_Independent,
                      numberOfThreadsPerBlock,
                      numberOfRepeats,
                      maxNumberOfCudaBlocks,
                      numberOfContractions,
                      maxNumberOfContractions,
                      contractionSize,
                      memorySize,
                      correctResults,
                      clearCacheStyle,
                      dev_junkDataToClearTheCache,
                      junkDataSize,
                      dev_contractionData_LayoutLeft_A,
                      dev_contractionData_LayoutLeft_B,
                      dev_junkDataCounter,
                      &totalNumberOfRepeats,
                      dev_contractionResults,
                      &contractionResults,
                      l,
                      r,
                      q,
                      i);

      }
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ***************** </do cuda independent> **********************
      // ===============================================================
      /*
      // ===============================================================
      // ***************** < do cuda reductions> ***********************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        const unsigned int numberOfThreadsPerBlock =
          std::min(unsigned(1024),
                   unsigned(ceil(dotProductSize / 32.)) * 32);

        cudaReduction_TimesMatrix[dotProductSizeIndex][memorySizeIndex] =
          runCudaTest(CudaStyle_Reduction,
                      numberOfThreadsPerBlock,
                      numberOfRepeats,
                      maxNumberOfCudaBlocks,
                      numberOfDotProducts,
                      maxNumberOfDotProducts,
                      dotProductSize,
                      memorySize,
                      correctResults,
                      clearCacheStyle,
                      dev_junkDataToClearTheCache,
                      junkDataSize,
                      dev_dotProductData_LayoutRight_A,
                      dev_dotProductData_LayoutRight_B,
                      dev_junkDataCounter,
                      &totalNumberOfRepeats,
                      dev_dotProductResults,
                      &dotProductResults);

      }
      cudaSwitchingTimesMatrix[dotProductSizeIndex][memorySizeIndex] =
        runSwitchingCudaTest(numberOfRepeats,
                             maxNumberOfCudaBlocks,
                             numberOfDotProducts,
                             maxNumberOfDotProducts,
                             dotProductSize,
                             memorySize,
                             correctResults,
                             clearCacheStyle,
                             dev_junkDataToClearTheCache,
                             junkDataSize,
                             dev_dotProductData_LayoutLeft_A,
                             dev_dotProductData_LayoutLeft_B,
                             dev_dotProductData_LayoutRight_A,
                             dev_dotProductData_LayoutRight_B,
                             dev_junkDataCounter,
                             &totalNumberOfRepeats,
                             dev_dotProductResults,
                             &dotProductResults);
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ***************** </do cuda reductions> ***********************
      // ===============================================================
*/
#ifdef ENABLE_KOKKOS
      // ===============================================================
      // ***************** < do kokkos> ********************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      /*
      {
        typedef Kokkos::OpenMP                             DeviceType;
        typedef Kokkos::View<float**, Kokkos::LayoutRight,
                             DeviceType>                   KokkosDotProductData;
        kokkosOmpTimesMatrix[dotProductSizeIndex][memorySizeIndex] =
          runKokkosTest<DeviceType,
                        KokkosDotProductData>(numberOfDotProducts,
                                              numberOfRepeats,
                                              dotProductSize,
                                              memorySize,
                                              dotProductData_LayoutRight_A,
                                              dotProductData_LayoutRight_B,
                                              correctResults,
                                              string("Kokkos openmp"),
                                              clearCacheStyle,
                                              junkDataToClearTheCache,
                                              &junkDataCounter,
                                              &totalNumberOfRepeats,
                                              &dotProductResults);
      } */
      {
        typedef Kokkos::Cuda                               DeviceType;
        typedef Kokkos::View<float****, Kokkos::LayoutLeft,
                             DeviceType>                   KokkosContractionData_Left;

        typedef Kokkos::View<float****, Kokkos::LayoutLeft,
                             DeviceType>                   KokkosContractionData_Right;
        // i pass in the layout right version even though this is the cuda
        //  version because it gets copied into the view inside the function.
        kokkosCudaIndependentTimesMatrix[contractionSizeIndex][memorySizeIndex] =
          runKokkosTest<DeviceType,
                        KokkosContractionData_Left,
                        KokkosContractionData_Right>(cellSize,
			                                        numberOfRepeats,
                                              memorySize,
                                              numCells,
                                              l,
                                              r,
                                              q,
                                              i,
                                              contractionData_LayoutRight_A,
                                              contractionData_LayoutRight_B,
                                              correctResults,
                                              string("Kokkos cuda"),
                                              clearCacheStyle,
                                              junkDataToClearTheCache,
                                              &junkDataCounter,
                                              &totalNumberOfRepeats,
                                              &contractionResults);
      }

      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ***************** </do kokkos> ********************************
      // ===============================================================
#endif // ENABLE_KOKKOS

      contractionSizeMatrix[contractionSizeIndex][memorySizeIndex] =
        contractionSize;
      numberOfContractionsMatrix[contractionSizeIndex][memorySizeIndex] =
        numCells;
      memorySizeMatrix[contractionSizeIndex][memorySizeIndex] =
        memorySize;

    }

    const timespec thisSizesToc = getTimePoint();
    const float thisSizesElapsedTime =
      getElapsedTime(thisSizesTic, thisSizesToc);
    printf("completed %4u repeats of dot products of size %4u "
           "in %7.2f seconds\n", numberOfRepeats,
           contractionSize, thisSizesElapsedTime);

    checkCudaError(hipFree(dev_contractionData_LayoutLeft_A));
    checkCudaError(hipFree(dev_contractionData_LayoutLeft_B));
    checkCudaError(hipFree(dev_contractionData_LayoutRight_A));
    checkCudaError(hipFree(dev_contractionData_LayoutRight_B));
    checkCudaError(hipFree(dev_contractionResults));

  }
  writeTimesMatrixToFile(contractionSizeMatrix,
                         prefix + string("contractionSize") + suffix);
  writeTimesMatrixToFile(numberOfContractionsMatrix,
                         prefix + string("numberOfContractions") + suffix);
  writeTimesMatrixToFile(memorySizeMatrix,
                         prefix + string("memorySize") + suffix);
  writeTimesMatrixToFile(serialTimesMatrix,
                         prefix + string("serialTimes") + suffix);
  writeTimesMatrixToFile(ompTimesMatrix,
                         prefix + string("ompTimes") + suffix);
  writeTimesMatrixToFile(cudaIndependent_TimesMatrix,
                         prefix + string("cudaIndependentTimes") + suffix);
  writeTimesMatrixToFile(cudaReduction_TimesMatrix,
                         prefix + string("cudaReductionTimes") + suffix);
  writeTimesMatrixToFile(cudaSwitchingTimesMatrix,
                         prefix + string("cudaSwitchingTimes") + suffix);
#ifdef ENABLE_KOKKOS
  writeTimesMatrixToFile(kokkosOmpTimesMatrix,
                         prefix + string("kokkosOmpTimes") + suffix);
  writeTimesMatrixToFile(kokkosCudaIndependentTimesMatrix,
                         prefix + string("kokkosCudaIndependentTimes") + suffix);
#endif

#ifdef ENABLE_KOKKOS
  const unsigned int numberOfMethods = 7;
#else
  const unsigned int numberOfMethods = 5;
#endif

  const size_t junkDataSum =
    std::accumulate(junkDataToClearTheCache.begin(),
                    junkDataToClearTheCache.end(), size_t(0));
  {
    int temp = 0;
    checkCudaError(hipMemcpy(&temp,
                              dev_junkDataCounter,
                              sizeof(int),
                              hipMemcpyDeviceToHost));
    junkDataCounter += temp;
  }
  if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
    const size_t expectedDataCounter = 0;
    if (junkDataCounter != expectedDataCounter) {
      fprintf(stderr, "for DontClearCacheAfterEveryRepeat, invalid "
              "junkDataCounter = %zu (%e), it should be %zu (%e)\n",
              junkDataCounter, float(junkDataCounter),
              expectedDataCounter, float(expectedDataCounter));
      exit(1);
    }
  } else {
    const size_t expectedDataCounter =
      junkDataSum * size_t(numberOfMethods) * (numberOfRepeats + 1) * numberOfMemorySizes *
      numberOfContractionSizes;
    if (junkDataCounter != expectedDataCounter) {
      fprintf(stderr, "for ClearCacheAfterEveryRepeat, invalid "
              "junkDataCounter = %zu (%e), it should be %zu (%e)\n",
              junkDataCounter, float(junkDataCounter),
              expectedDataCounter, float(expectedDataCounter));
      exit(1);
    }
  }

  const unsigned int expectedTotalNumberOfRepeats = numberOfMethods *
    (numberOfRepeats + 1) * numberOfMemorySizes * numberOfContractionSizes;
  if (totalNumberOfRepeats != expectedTotalNumberOfRepeats) {
    fprintf(stderr, "invalid totalNumberOfRepeats = %u (%e), it should be "
            "%u (%e)\n",
            totalNumberOfRepeats, float(totalNumberOfRepeats),
            expectedTotalNumberOfRepeats, float(expectedTotalNumberOfRepeats));
    exit(1);
  }

#ifdef ENABLE_KOKKOS
  Kokkos::finalize();
#endif

  return 0;
}
