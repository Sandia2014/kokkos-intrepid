#include "hip/hip_runtime.h"
// -*- C++ -*-
// ContractDataFieldScalar.cu
// a huge comparison of different ways of doing ContractDataFieldScalar
// Tyler Marklyn (outline stolen from Jeff Amelang), 2015

// c junk
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <cstdio>
#include <unistd.h>

// c++ junk
#include <vector>
#include <algorithm>
#include <chrono>
#include <random>
#include <string>
#include <array>
#include <fstream>
using std::string;
using std::vector;
using std::array;

// header file for openmp
#include <omp.h>

#define ENABLE_KOKKOS
#ifdef ENABLE_KOKKOS
#include <Kokkos_Core.hpp>
#include "ContractDataFieldScalarFunctors.hpp"
#endif // ENABLE_KOKKOS

enum ClearCacheStyle {ClearCacheAfterEveryRepeat,
                      DontClearCacheAfterEveryRepeat};

// stolen from http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline
void
gpuAssert(const hipError_t code, const char *file, const int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPU Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort == true) {
      exit(code);
    }
  }
}

timespec
getTimePoint() {
  timespec timepoint;
  clock_gettime(CLOCK_MONOTONIC, &timepoint);
  return timepoint;
}

// yay for having to use pre-c++11 timing because of nvcc
double
getElapsedTime(const timespec & start, const timespec & end) {
  timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}


void
writeTimesMatrixToFile(const vector<vector<float> > & times,
                       const string filename) {

  const unsigned int numberOfContractionSizes = times.size();
  // yeah, yeah, kinda unsafe
  const unsigned int numberOfMemorySizes = times[0].size();
  char sprintfBuffer[500];
  sprintf(sprintfBuffer, "%s.csv", filename.c_str());
  FILE* file = fopen(sprintfBuffer, "w");
  for (unsigned int contractionSizeIndex = 0;
       contractionSizeIndex < numberOfContractionSizes;
       ++contractionSizeIndex) {
    for (unsigned int memorySizeIndex = 0;
         memorySizeIndex < numberOfMemorySizes;
         ++memorySizeIndex) {
      if (memorySizeIndex > 0) {
        fprintf(file, ", ");
      }
      fprintf(file, "%10.4e", times[contractionSizeIndex][memorySizeIndex]);
    }
    fprintf(file, "\n");
  }
  fclose(file);
}

void
checkAnswer(const vector<float> & correctResults,
            const vector<float> & calcResults,
            const unsigned int contractionSize,
            const unsigned int memorySize,
            const string flavorName) {
  for (unsigned int dotProductIndex = 0;
       dotProductIndex < correctResults.size();
       ++dotProductIndex) {
    if (std::abs(correctResults[dotProductIndex] -
                 calcResults[dotProductIndex]) /
        std::abs(correctResults[dotProductIndex]) > 1e-4) {
      fprintf(stderr, "invalid answer for dot product index %u for "
              "flavor %s, "
              "should be %e but we have %e, "
              "contractionSize = %u, memorySize = %8.2e\n",
              dotProductIndex, flavorName.c_str(),
              correctResults[dotProductIndex],
              calcResults[dotProductIndex],
              contractionSize, float(memorySize));
      exit(1);
    }
  }
}


#ifdef ENABLE_KOKKOS


template <class DeviceType, class KokkosInputData, class KokkosInputField>
double
runKokkosTest(const unsigned int numberOfRepeats,
              const unsigned int memorySize,
              const unsigned int numCells,
              const unsigned int numPoints,
              const unsigned int numFields,
              const vector<float> & inputData_LayoutRight,
              const vector<float> & inputField_LayoutRight,
              const vector<float> & correctResults,
              const string & kokkosFlavor,
              const ClearCacheStyle clearCacheStyle,
              const vector<int> & junkDataToClearTheCache,
              size_t * junkDataCounter,
              unsigned int * const totalNumberOfRepeats,
              vector<float> * calcResults) {

  const unsigned int junkDataSize = junkDataToClearTheCache.size();

  typedef typename KokkosInputField::HostMirror          KokkosInputField_Host;
  typedef typename KokkosInputData::HostMirror          KokkosInputData_Host;

  typedef Kokkos::View<float**, DeviceType>              KokkosCalcResults;
  typedef typename KokkosCalcResults::HostMirror        KokkosCalcResults_Host;
  typedef Kokkos::View<int*, DeviceType>                KokkosJunkVector;
  typedef typename KokkosJunkVector::HostMirror         KokkosJunkVector_Host;

  KokkosInputData dev_kokkosInputData_A("kokkos data A",
                                                  numCells, numPoints);
  KokkosInputData_Host kokkosInputData_A =
    Kokkos::create_mirror_view(dev_kokkosInputData_A);

  KokkosInputField dev_kokkosInputField("kokkos data B",
                                                  numCells, numFields, numPoints);
  KokkosInputField_Host kokkosInputField =
    Kokkos::create_mirror_view(dev_kokkosInputField);

  KokkosCalcResults dev_kokkosCalcResults("kokkos dot product results",
                                                      numCells, numFields);
  KokkosCalcResults_Host kokkosCalcResults =
    Kokkos::create_mirror_view(dev_kokkosCalcResults);

  KokkosJunkVector dev_kokkosJunkDataToClearTheCache("kokkos junk data to clear cache",
                                                     junkDataSize);
  KokkosJunkVector_Host kokkosJunkDataToClearTheCache =
    Kokkos::create_mirror_view(dev_kokkosJunkDataToClearTheCache);

  // copy the data into the device views and ship them over
  for (int cl = 0; cl < numCells; ++cl) {
    for (int qp = 0; qp < numPoints; ++qp) {
          kokkosInputData_A(cl, qp) =
            inputData_LayoutRight[cl * numPoints + qp];
      for (int lbf = 0; lbf < numFields; ++lbf) {
          kokkosInputField(cl, lbf, qp) =
            inputField_LayoutRight[cl * numPoints * numFields + lbf * numPoints + qp];
      }
    }
  }

  Kokkos::deep_copy(dev_kokkosInputData_A, kokkosInputData_A);
  Kokkos::deep_copy(dev_kokkosInputField, kokkosInputField);

  // copy the data into the device views and ship them over
  for (unsigned int junkDataIndex = 0;
       junkDataIndex < junkDataSize; ++junkDataIndex) {
    kokkosJunkDataToClearTheCache(junkDataIndex) =
      junkDataToClearTheCache[junkDataIndex];
  }
  Kokkos::deep_copy(dev_kokkosJunkDataToClearTheCache, kokkosJunkDataToClearTheCache);

  KokkosFunctor_ClearCache<DeviceType,
                           KokkosJunkVector>
    kokkosFunctor_ClearCache(dev_kokkosJunkDataToClearTheCache);

  // breaking formatting convention because holy freak that's long
  ContractDataFieldScalarFunctor<DeviceType,
                            KokkosInputData,
                            KokkosInputField,
                            KokkosCalcResults>
    contractDataFieldScalarFunctor(numPoints,
                              numFields,
                              dev_kokkosInputField,
                              dev_kokkosInputData_A,
                              dev_kokkosCalcResults);

  timespec tic;
  double totalElapsedTime = 0;
  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
    *totalNumberOfRepeats = *totalNumberOfRepeats + 1;
    if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
         repeatIndex == 1) ||
        clearCacheStyle == ClearCacheAfterEveryRepeat) {
      tic = getTimePoint();
    }

    // actually do the calculation
    Kokkos::parallel_for(numCells, contractDataFieldScalarFunctor);

    // wait for this repeat's results to finish
    Kokkos::fence();

    if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
      const timespec toc = getTimePoint();
      const float elapsedTime = getElapsedTime(tic, toc);
      totalElapsedTime += elapsedTime;

      // attempt to scrub all levels of cache
      size_t partialJunkDataCounter = 0;
      Kokkos::parallel_reduce(junkDataSize, kokkosFunctor_ClearCache,
                              partialJunkDataCounter);
      *junkDataCounter += partialJunkDataCounter;
    }
  }
  if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
    const timespec toc = getTimePoint();
    totalElapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
  }
  // copy over the results from the device to the host
  Kokkos::deep_copy(kokkosCalcResults, dev_kokkosCalcResults);
  for (unsigned int dotProductIndex = 0;
       dotProductIndex < numCells; ++dotProductIndex) {
    for (unsigned int lbf = 0; lbf < numFields; ++lbf) {
      calcResults->at(dotProductIndex * numFields + lbf) =
      kokkosCalcResults(dotProductIndex, lbf);
    }
  }
  // check the results
  checkAnswer(correctResults, *calcResults,
              numPoints * numFields , memorySize,
              kokkosFlavor);

  // scrub the results
  std::fill(calcResults->begin(),
            calcResults->end(),
            std::numeric_limits<float>::quiet_NaN());

  return totalElapsedTime;
}

#endif // ENABLE_KOKKOS



int main(int argc, char* argv[]) {

#ifdef ENABLE_KOKKOS
  Kokkos::initialize(argc, argv);
#endif

  // ===============================================================
  // ********************** < input> ******************************
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
  const vector<unsigned int> contractionSizes =
    {{25, 100, 500, 1000, 2000}};
    //{{8, 16, 32, 64, 128, 256, 512, 1024, 2048}};
  const array<float, 2> memorySizeExtrema = {{1e6, 1e9}};
  const unsigned int numberOfMemorySizes = 20;
  const unsigned int numFields = 5;


  const ClearCacheStyle clearCacheStyle =
    ClearCacheAfterEveryRepeat;
  const unsigned int numberOfRepeats =
    (clearCacheStyle == ClearCacheAfterEveryRepeat) ? 10 : 250;
  const string machineName = "shadowfax";
  const string prefix = "data/ContractDataFieldScalar_";
  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  // ********************** </input> ******************************
  // ===============================================================

  // derive some values from the inputs
  const unsigned int numberOfContractionSizes = contractionSizes.size();
  const string clearCacheStyleString =
    (clearCacheStyle == ClearCacheAfterEveryRepeat) ? "clearCache" :
    "dontClearCache";
  const string suffix = "_" + clearCacheStyleString + "_" + machineName;

  // create the actual sizes
  vector<unsigned int> memorySizes(numberOfMemorySizes);
  for (unsigned int memorySizeIndex = 0;
       memorySizeIndex < numberOfMemorySizes; ++memorySizeIndex) {
    const float percent = memorySizeIndex / float(numberOfMemorySizes - 1);
    const float minLog = log10(memorySizeExtrema[0]);
    const float maxLog = log10(memorySizeExtrema[1]);
    const float thisLog = minLog + percent * (maxLog - minLog);
    const unsigned int maxContractionSize = contractionSizes.back();
    // memory size is linear on a log scale, but rounded to a multiple of the
    //  largest dot product size
    const unsigned int desiredMemorySizeInBytes = pow(10., thisLog);
    // now, in this amount of memory i have to fit two vectors of data
    // that are multiples of the max dot product size
    const unsigned int memorySizeInBytes =
      unsigned(desiredMemorySizeInBytes /
               float(4 * sizeof(float) * maxContractionSize)) *
      4 * sizeof(float) * maxContractionSize;
    memorySizes[memorySizeIndex] = memorySizeInBytes;
  }

  // create a c++11 random number generator
  std::mt19937 randomNumberEngine;
  std::uniform_real_distribution<float> randomNumberGenerator(0, 1);

  // these are just containers for storing the numbers we'll be plotting.
  // i feel a little dirty using a vector<vector>, but i don't want to introduce
  //  a dependence on eigen or something for a real matrix.
  vector<vector<float> >
    contractionSizeMatrix(numberOfContractionSizes,
                         vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    numberOfDotProductsMatrix(numberOfContractionSizes,
                              vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    memorySizeMatrix(numberOfContractionSizes,
                     vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    serialTimesMatrix(numberOfContractionSizes,
                      vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    ompTimesMatrix(numberOfContractionSizes,
                   vector<float>(numberOfMemorySizes, 0));

#ifdef ENABLE_KOKKOS
  vector<vector<float> >
    kokkosOmpTimesMatrix(numberOfContractionSizes,
                         vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    kokkosCudaIndependentTimesMatrix(numberOfContractionSizes,
                                     vector<float>(numberOfMemorySizes, 0));
#endif

  // create some junk data to use in clearing the cache
  size_t junkDataCounter = 0;
  const size_t junkDataSize = 1e7;
  vector<int> junkDataToClearTheCache(junkDataSize, 0);
  for (unsigned int i = 0; i < junkDataSize/100; ++i) {
    junkDataToClearTheCache[(rand() / float(RAND_MAX))*junkDataSize] = 1;
  }


  unsigned int totalNumberOfRepeats = 0;

  // for each dot product size
  for (unsigned int contractionSizeIndex = 0;
       contractionSizeIndex < numberOfContractionSizes;
       ++contractionSizeIndex) {
    const unsigned int contractionSize = contractionSizes[contractionSizeIndex];
    //const unsigned int dimVec = 8;
    const unsigned int numPoints = contractionSize / numFields;

    const timespec thisSizesTic = getTimePoint();

    // allocate and initialize the largest amount of memory we'll need, then on
    //  each size we'll just use subsets of this memory.
    const unsigned int maxNumberOfDotProducts =
      memorySizes.back() / 4 / sizeof(float) / contractionSize;
    vector<float> inputData_LayoutRight(maxNumberOfDotProducts * contractionSize);
    vector<float> inputField_LayoutRight(maxNumberOfDotProducts * contractionSize);
    //vector<float> inputField_LayoutRight(inputData_LayoutRight.size());
    vector<float> dotProductData_LayoutLeft_A(inputData_LayoutRight.size());
    vector<float> dotProductData_LayoutLeft_B(inputField_LayoutRight.size());

    for (unsigned int dotProductIndex = 0;
         dotProductIndex < maxNumberOfDotProducts; ++dotProductIndex) {
      for (unsigned int entryIndex = 0;
           entryIndex < contractionSize; ++entryIndex) {

        const unsigned int layoutRightIndex =
          dotProductIndex * contractionSize + entryIndex;
        inputData_LayoutRight[layoutRightIndex] =
          randomNumberGenerator(randomNumberEngine);
        inputField_LayoutRight[layoutRightIndex] =
          randomNumberGenerator(randomNumberEngine);

        const unsigned int layoutLeftIndex =
          entryIndex * maxNumberOfDotProducts + dotProductIndex;
        dotProductData_LayoutLeft_A[layoutLeftIndex] =
          inputData_LayoutRight[layoutRightIndex];
        dotProductData_LayoutLeft_B[layoutLeftIndex] =
          inputField_LayoutRight[layoutRightIndex];
      }
    }
    vector<float> calcResults(maxNumberOfDotProducts * numFields,
                                    std::numeric_limits<float>::quiet_NaN());

    // for each memory size
    for (unsigned int memorySizeIndex = 0;
         memorySizeIndex < numberOfMemorySizes;
         ++memorySizeIndex) {
      const unsigned int memorySize = memorySizes[memorySizeIndex];


      const unsigned int numCells =
        memorySize / 4 / sizeof(float) / contractionSize;
      if (memorySize != 4 * sizeof(float) * numCells * contractionSize) {
        fprintf(stderr, "invalid memory size of %u for dot product size of "
                "%u because it doesn't divide evenly, remainder is %zu\n",
                memorySize, contractionSize,
                memorySize % (4 * sizeof(float) * contractionSize));
        exit(1);
      }

      // ===============================================================
      // ********************** < do serial> ***************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        timespec tic;
        for (unsigned int repeatIndex = 0;
            repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
          ++totalNumberOfRepeats;
          if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
                repeatIndex == 1) ||
              clearCacheStyle == ClearCacheAfterEveryRepeat) {
            tic = getTimePoint();
          }

          for (int cl = 0; cl < numCells; cl++) {
            for (int lbf = 0; lbf < numFields; lbf++) {
              double tmpVal = 0;
              for (int qp = 0; qp < numPoints; qp++) {
                tmpVal += inputField_LayoutRight[cl * numPoints *numFields + lbf * numPoints + qp] *
                  inputData_LayoutRight[cl * numPoints +  qp];
              } // P-loop
              calcResults[cl * numFields + lbf] = tmpVal;
            } // F-loop
          } // C-loop

          if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
            const timespec toc = getTimePoint();
            const float elapsedTime = getElapsedTime(tic, toc);
            serialTimesMatrix[contractionSizeIndex][memorySizeIndex] += elapsedTime;

            junkDataCounter +=
              std::accumulate(junkDataToClearTheCache.begin(),
                              junkDataToClearTheCache.end(), size_t(0));
          }
        }
        if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
          const timespec toc = getTimePoint();
          const float elapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
          serialTimesMatrix[contractionSizeIndex][memorySizeIndex] = elapsedTime;
        }
      }
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ********************** </do serial> ***************************
      // ===============================================================

      const vector<float> correctResults = calcResults;
      // scrub the results
      std::fill(calcResults.begin(),
                calcResults.end(),
                std::numeric_limits<float>::quiet_NaN());

      // ===============================================================
      // ********************** < do omp> ******************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        timespec tic;
        for (unsigned int repeatIndex = 0;
             repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
          ++totalNumberOfRepeats;
          if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
               repeatIndex == 1) ||
              clearCacheStyle == ClearCacheAfterEveryRepeat) {
            tic = getTimePoint();
          }

          // do the actual calculation
#pragma omp parallel for default(none)                                  \
  shared(inputData_LayoutRight, inputField_LayoutRight,    \
         calcResults)


          for (int cl = 0; cl < numCells; cl++) {
            for (int lbf = 0; lbf < numFields; lbf++) {
              double tmpVal = 0;
              for (int qp = 0; qp < numPoints; qp++) {
                tmpVal += inputField_LayoutRight[cl * numPoints *numFields + lbf * numPoints + qp] *
                  inputData_LayoutRight[cl * numPoints +  qp];
              } // P-loop
              calcResults[cl * numFields + lbf] = tmpVal;
            } // F-loop
          } // C-loop

          if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
            const timespec toc = getTimePoint();
            const float elapsedTime = getElapsedTime(tic, toc);
            ompTimesMatrix[contractionSizeIndex][memorySizeIndex] += elapsedTime;

            // attempt to scrub all levels of cache
#pragma omp parallel default(none)                      \
  shared(junkDataCounter, junkDataToClearTheCache)
            {
              const size_t thisThreadsJunkDataCounter =
                std::accumulate(junkDataToClearTheCache.begin(),
                                junkDataToClearTheCache.end(), size_t(0));
              // only one thread adds the junk counter so that the total
              //  at the end is not a function of the number of threads.
#pragma omp single
              junkDataCounter += thisThreadsJunkDataCounter;
            }
          }
        }
        if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
          const timespec toc = getTimePoint();
          const float elapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
          ompTimesMatrix[contractionSizeIndex][memorySizeIndex] = elapsedTime;
          // check the results
          checkAnswer(correctResults, calcResults,
                      contractionSize * numFields, memorySize,
                      string("omp"));
        }
      }
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ********************** </do omp> ******************************
      // ===============================================================

      // scrub the results
      std::fill(calcResults.begin(),
                calcResults.end(),
                std::numeric_limits<float>::quiet_NaN());

#ifdef ENABLE_KOKKOS
      // ===============================================================
      // ***************** < do kokkos> ********************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

      {
        typedef Kokkos::OpenMP                             DeviceType;
        typedef Kokkos::View<float**, Kokkos::LayoutRight,
                             DeviceType>                   KokkosInputData;
        typedef Kokkos::View<float***, Kokkos::LayoutRight,
                             DeviceType>                   KokkosInputField;
        kokkosOmpTimesMatrix[contractionSizeIndex][memorySizeIndex] =
          runKokkosTest<DeviceType,
                        KokkosInputData,
                        KokkosInputField>(numberOfRepeats,
                                              memorySize,
                                              numCells,
                                              numPoints,
                                              numFields,
                                              inputData_LayoutRight,
                                              inputField_LayoutRight,
                                              correctResults,
                                              string("Kokkos openmp"),
                                              clearCacheStyle,
                                              junkDataToClearTheCache,
                                              &junkDataCounter,
                                              &totalNumberOfRepeats,
                                              &calcResults);
      }
      {
        typedef Kokkos::Cuda                               DeviceType;
        typedef Kokkos::View<float**, Kokkos::LayoutLeft,
                             DeviceType>                   KokkosInputData;
        typedef Kokkos::View<float***, Kokkos::LayoutLeft,
                             DeviceType>                   KokkosInputField;
        // i pass in the layout right version even though this is the cuda
        //  version because it gets copied into the view inside the function.
        kokkosCudaIndependentTimesMatrix[contractionSizeIndex][memorySizeIndex] =
          runKokkosTest<DeviceType,
                        KokkosInputData,
                         KokkosInputField >(numberOfRepeats,
                                              memorySize,
                                              numCells,
                                              numPoints,
                                              numFields,
                                              inputData_LayoutRight,
                                              inputField_LayoutRight,
                                              correctResults,
                                              string("Kokkos cuda"),
                                              clearCacheStyle,
                                              junkDataToClearTheCache,
                                              &junkDataCounter,
                                              &totalNumberOfRepeats,
                                              &calcResults);
      }

      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ***************** </do kokkos> ********************************
      // ===============================================================
#endif // ENABLE_KOKKOS

      contractionSizeMatrix[contractionSizeIndex][memorySizeIndex] =
        contractionSize;
      numberOfDotProductsMatrix[contractionSizeIndex][memorySizeIndex] =
        numCells;
      memorySizeMatrix[contractionSizeIndex][memorySizeIndex] =
        memorySize;

    }

    const timespec thisSizesToc = getTimePoint();
    const float thisSizesElapsedTime =
      getElapsedTime(thisSizesTic, thisSizesToc);
    printf("completed %4u repeats of dot products of size %4u "
           "in %7.2f seconds\n", numberOfRepeats,
           contractionSize, thisSizesElapsedTime);


  }
  writeTimesMatrixToFile(contractionSizeMatrix,
                         prefix + string("contractionSize") + suffix);
  writeTimesMatrixToFile(numberOfDotProductsMatrix,
                         prefix + string("numberOfDotProducts") + suffix);
  writeTimesMatrixToFile(memorySizeMatrix,
                         prefix + string("memorySize") + suffix);
  writeTimesMatrixToFile(serialTimesMatrix,
                         prefix + string("serialTimes") + suffix);
  writeTimesMatrixToFile(ompTimesMatrix,
                         prefix + string("ompTimes") + suffix);


#ifdef ENABLE_KOKKOS
  writeTimesMatrixToFile(kokkosOmpTimesMatrix,
                         prefix + string("kokkosOmpTimes") + suffix);
  writeTimesMatrixToFile(kokkosCudaIndependentTimesMatrix,
                         prefix + string("kokkosCudaIndependentTimes") + suffix);
#endif

#if defined RAW_CUDA
  // Note, we assume that if RAW_CUDA is defined so is ENABLE_KOKKOS here
  const unsigned int numberOfMethods = 7;
#elif defined ENABLE_KOKKOS
  const unsigned int numberOfMethods = 4;
#else
  const unsigned int numberOfMethods = 2;
#endif

  const size_t junkDataSum =
    std::accumulate(junkDataToClearTheCache.begin(),
                    junkDataToClearTheCache.end(), size_t(0));
  {
    int temp = 0;
    junkDataCounter += temp;
  }
  if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
    const size_t expectedDataCounter = 0;
    if (junkDataCounter != expectedDataCounter) {
      fprintf(stderr, "for DontClearCacheAfterEveryRepeat, invalid "
              "junkDataCounter = %zu (%e), it should be %zu (%e)\n",
              junkDataCounter, float(junkDataCounter),
              expectedDataCounter, float(expectedDataCounter));
      exit(1);
    }
  } else {
    const size_t expectedDataCounter =
      junkDataSum * size_t(numberOfMethods) * (numberOfRepeats + 1) * numberOfMemorySizes *
      numberOfContractionSizes;
    if (junkDataCounter != expectedDataCounter) {
      fprintf(stderr, "for ClearCacheAfterEveryRepeat, invalid "
              "junkDataCounter = %zu (%e), it should be %zu (%e)\n",
              junkDataCounter, float(junkDataCounter),
              expectedDataCounter, float(expectedDataCounter));
      exit(1);
    }
  }

  const unsigned int expectedTotalNumberOfRepeats = numberOfMethods *
    (numberOfRepeats + 1) * numberOfMemorySizes * numberOfContractionSizes;
  if (totalNumberOfRepeats != expectedTotalNumberOfRepeats) {
    fprintf(stderr, "invalid totalNumberOfRepeats = %u (%e), it should be "
            "%u (%e)\n",
            totalNumberOfRepeats, float(totalNumberOfRepeats),
            expectedTotalNumberOfRepeats, float(expectedTotalNumberOfRepeats));
    exit(1);
  }

#ifdef ENABLE_KOKKOS
  Kokkos::finalize();
#endif

  return 0;
}
