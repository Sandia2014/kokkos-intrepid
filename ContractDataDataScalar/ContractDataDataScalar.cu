#include "hip/hip_runtime.h"
// -*- C++ -*-
// ArrayOfDotProducts.cc
// a huge comparison of different ways of doing an array of dot products
// Jeff Amelang, 2014

// c junk
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <cstdio>
#include <unistd.h>

// c++ junk
#include <vector>
#include <algorithm>
#include <chrono>
#include <random>
#include <string>
#include <array>
#include <fstream>
using std::string;
using std::vector;
using std::array;

// header file for openmp
#include <omp.h>

#ifdef ENABLE_KOKKOS
#include <Kokkos_Core.hpp>
#include "ContractDataDataScalarFunctors.hpp"
#endif // ENABLE_KOKKOS

enum CudaStyle {CudaStyle_Independent,
                CudaStyle_Reduction};

enum ClearCacheStyle {ClearCacheAfterEveryRepeat,
                      DontClearCacheAfterEveryRepeat};

string
convertCudaStyleToString(const CudaStyle cudaStyle) {
  switch (cudaStyle) {
  case CudaStyle_Independent:
    return string("CudaStyle_Independent");
  case CudaStyle_Reduction:
    return string("CudaStyle_Reduction");
  default:
    fprintf(stderr, "invalid cuda style\n");
    exit(1);
  };
}

// stolen from http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline
void
gpuAssert(const hipError_t code, const char *file, const int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPU Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort == true) {
      exit(code);
    }
  }
}

timespec
getTimePoint() {
  timespec timepoint;
  clock_gettime(CLOCK_MONOTONIC, &timepoint);
  return timepoint;
}

// yay for having to use pre-c++11 timing because of nvcc
double
getElapsedTime(const timespec & start, const timespec & end) {
  timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}





__global__
void
doCudaClearCache_kernel(const unsigned int junkDataSize,
                        const int * const __restrict__ dev_junkDataToClearTheCache,
                        int * dev_result) {
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  int partialSum = 0;
  while (index < junkDataSize) {
    partialSum += dev_junkDataToClearTheCache[index];
    index += blockDim.x * gridDim.x;
  }
  atomicAdd(dev_result, partialSum);
}

__global__
void
doCudaDotProducts_Independent_kernel(const unsigned int numberOfDotProducts,
                                     const unsigned int maxNumberOfDotProducts,
                                     const unsigned int dotProductSize,
                                     const float * const __restrict__ dev_dotProductData_LayoutLeft_A,
                                     const float * const __restrict__ dev_dotProductData_LayoutLeft_B,
                                     float * dev_dotProductResults) {
  unsigned int dotProductIndex = blockIdx.x * blockDim.x + threadIdx.x;
  while (dotProductIndex < numberOfDotProducts) {
    float sum = 0;
    for (unsigned int entryIndex = 0; entryIndex < dotProductSize; ++entryIndex) {
      const unsigned int index = dotProductIndex +
        entryIndex * maxNumberOfDotProducts;
      sum +=
        dev_dotProductData_LayoutLeft_A[index] *
        dev_dotProductData_LayoutLeft_B[index];
    }
    dev_dotProductResults[dotProductIndex] = sum;
    dotProductIndex += blockDim.x * gridDim.x;
  }
}

__global__
void
doCudaDotProducts_Reduction_kernel(const unsigned int numberOfDotProducts,
                                   const unsigned int dotProductSize,
                                   const float * const __restrict__ dev_dotProductData_LayoutRight_A,
                                   const float * const __restrict__ dev_dotProductData_LayoutRight_B,
                                   float * dev_dotProductResults) {

  extern __shared__ float sharedMemory[];

  unsigned int dotProductIndex = blockIdx.x;
  while (dotProductIndex < numberOfDotProducts) {

    // goal: compute the contribution to the dot product from this thread
    const unsigned int shortcutIndex = dotProductIndex * dotProductSize;
    float partialSum = 0;
    unsigned int entryIndex = threadIdx.x;
    while (entryIndex < dotProductSize) {
      const unsigned int index = shortcutIndex + entryIndex;
      partialSum +=
        dev_dotProductData_LayoutRight_A[index] *
        dev_dotProductData_LayoutRight_B[index];
      entryIndex += blockDim.x;
    }
    // set this thread's value
    sharedMemory[threadIdx.x] = partialSum;

    // goal: reduce the warp's contribution to one number and add it to the
    //  dot product.

    // each warp does its own reduction
    const unsigned int warpIndex = threadIdx.x / 32;
    const unsigned int indexWithinWarp = threadIdx.x - warpIndex * 32;
    const unsigned int warpStartingIndexInSharedMemory = warpIndex * 32;
    // the first quarter of the threads in the warp make small partial sums
    if (indexWithinWarp < 8) {
      const int i = indexWithinWarp * 4;
      partialSum  = sharedMemory[warpStartingIndexInSharedMemory + i];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + i + 1];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + i + 2];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + i + 3];
      sharedMemory[warpStartingIndexInSharedMemory + i] = partialSum;
    }
    // the first thread in the warp reduces the 8 partial sums
    if (indexWithinWarp == 0) {
      partialSum += sharedMemory[warpStartingIndexInSharedMemory +  4];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory +  8];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 12];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 16];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 20];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 24];
      partialSum += sharedMemory[warpStartingIndexInSharedMemory + 28];
      // and adds it to the global sum
      atomicAdd(&dev_dotProductResults[dotProductIndex], partialSum);
    }

    // move on to the next dot product
    dotProductIndex += gridDim.x;
  }
}

void
writeTimesMatrixToFile(const vector<vector<float> > & times,
                       const string filename) {

  const unsigned int numberOfDotProductSizes = times.size();
  // yeah, yeah, kinda unsafe
  const unsigned int numberOfMemorySizes = times[0].size();
  char sprintfBuffer[500];
  sprintf(sprintfBuffer, "%s.csv", filename.c_str());
  FILE* file = fopen(sprintfBuffer, "w");
  for (unsigned int dotProductSizeIndex = 0;
       dotProductSizeIndex < numberOfDotProductSizes;
       ++dotProductSizeIndex) {
    for (unsigned int memorySizeIndex = 0;
         memorySizeIndex < numberOfMemorySizes;
         ++memorySizeIndex) {
      if (memorySizeIndex > 0) {
        fprintf(file, ", ");
      }
      fprintf(file, "%10.4e", times[dotProductSizeIndex][memorySizeIndex]);
    }
    fprintf(file, "\n");
  }
  fclose(file);
}

void
checkAnswer(const vector<float> & correctResults,
            const vector<float> & dotProductResults,
            const unsigned int dotProductSize,
            const unsigned int memorySize,
            const string flavorName) {
  for (unsigned int dotProductIndex = 0;
       dotProductIndex < correctResults.size();
       ++dotProductIndex) {
    if (std::abs(correctResults[dotProductIndex] -
                 dotProductResults[dotProductIndex]) /
        std::abs(correctResults[dotProductIndex]) > 1e-4) {
      fprintf(stderr, "invalid answer for dot product index %u for "
              "flavor %s, "
              "should be %e but we have %e, "
              "dotProductSize = %u, memorySize = %8.2e\n",
              dotProductIndex, flavorName.c_str(),
              correctResults[dotProductIndex],
              dotProductResults[dotProductIndex],
              dotProductSize, float(memorySize));
      exit(1);
    }
  }
}

double
runCudaTest(const CudaStyle cudaStyle,
            const unsigned int numberOfThreadsPerBlock,
            const unsigned int numberOfRepeats,
            const unsigned int maxNumberOfCudaBlocks,
            const unsigned int numberOfDotProducts,
            const unsigned int maxNumberOfDotProducts,
            const unsigned int dotProductSize,
            const unsigned int memorySize,
            const vector<float> & correctResults,
            const ClearCacheStyle clearCacheStyle,
            const int * const dev_junkDataToClearTheCache,
            const unsigned int junkDataSize,
            const float * const dev_dotProductData_A,
            const float * const dev_dotProductData_B,
            int * const dev_junkDataCounter,
            unsigned int * const totalNumberOfRepeats,
            float * const dev_dotProductResults,
            vector<float> * const dotProductResults) {
  const unsigned int numberOfBlocks =
    min(maxNumberOfCudaBlocks,
        (unsigned int)ceil(numberOfDotProducts/float(numberOfThreadsPerBlock)));

  timespec tic;
  double totalElapsedTime = 0;
  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
    *totalNumberOfRepeats = *totalNumberOfRepeats + 1;
    if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
         repeatIndex == 1) ||
        clearCacheStyle == ClearCacheAfterEveryRepeat) {
      tic = getTimePoint();
    }

    // do the actual calculation
    if (cudaStyle == CudaStyle_Independent) {
      doCudaDotProducts_Independent_kernel<<<numberOfBlocks,
        numberOfThreadsPerBlock>>>(numberOfDotProducts,
                                   maxNumberOfDotProducts,
                                   dotProductSize,
                                   dev_dotProductData_A,
                                   dev_dotProductData_B,
                                   dev_dotProductResults);
    } else if (cudaStyle == CudaStyle_Reduction) {
      doCudaDotProducts_Reduction_kernel<<<numberOfBlocks,
        numberOfThreadsPerBlock,
        numberOfThreadsPerBlock * sizeof(float)>>>(numberOfDotProducts,
                                                   dotProductSize,
                                                   dev_dotProductData_A,
                                                   dev_dotProductData_B,
                                                   dev_dotProductResults);
    } else {
      fprintf(stderr, "unknown cuda style\n");
      exit(1);
    }

    // wait for the kernel launch
    checkCudaError(hipPeekAtLastError());
    checkCudaError(hipDeviceSynchronize());
    if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
      const timespec toc = getTimePoint();
      const float elapsedTime = getElapsedTime(tic, toc);
      totalElapsedTime += elapsedTime;

      const unsigned int junkNumberOfBlocks =
        min(maxNumberOfCudaBlocks,
            (unsigned int)ceil(junkDataSize/float(numberOfThreadsPerBlock)));
      doCudaClearCache_kernel<<<junkNumberOfBlocks,
        numberOfThreadsPerBlock>>>(junkDataSize,
                                   dev_junkDataToClearTheCache,
                                   dev_junkDataCounter);
      // wait for the kernel launch
      checkCudaError(hipPeekAtLastError());
      checkCudaError(hipDeviceSynchronize());
    }
  }
  if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
    const timespec toc = getTimePoint();
    const float elapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
    totalElapsedTime = elapsedTime;
  }
  // copy over the results from the gpu to the cpu
  checkCudaError(hipMemcpy(&dotProductResults->at(0), dev_dotProductResults,
                            numberOfDotProducts * sizeof(float),
                            hipMemcpyDeviceToHost));
  // check the results
  checkAnswer(correctResults, *dotProductResults,
              dotProductSize, memorySize,
              convertCudaStyleToString(cudaStyle));

  // scrub the results
  std::fill(dotProductResults->begin(),
            dotProductResults->end(),
            std::numeric_limits<float>::quiet_NaN());
  checkCudaError(hipMemcpy(dev_dotProductResults, &dotProductResults->at(0),
                            numberOfDotProducts * sizeof(float),
                            hipMemcpyHostToDevice));

  return totalElapsedTime;
}

double
runSwitchingCudaTest(const unsigned int numberOfRepeats,
                     const unsigned int maxNumberOfCudaBlocks,
                     const unsigned int numberOfDotProducts,
                     const unsigned int maxNumberOfDotProducts,
                     const unsigned int dotProductSize,
                     const unsigned int memorySize,
                     const vector<float> & correctResults,
                     const ClearCacheStyle clearCacheStyle,
                     const int * const dev_junkDataToClearTheCache,
                     const unsigned int junkDataSize,
                     const float * const dev_dotProductData_LayoutLeft_A,
                     const float * const dev_dotProductData_LayoutLeft_B,
                     const float * const dev_dotProductData_LayoutRight_A,
                     const float * const dev_dotProductData_LayoutRight_B,
                     int * const dev_junkDataCounter,
                     unsigned int * const totalNumberOfRepeats,
                     float * const dev_dotProductResults,
                     vector<float> * const dotProductResults) {
  // if i can't saturate occupancy, do the reduction version
  // i got this number by just looking at where the plots crossed, where
  //  the reduction style actually starts beating the independent.
  if (numberOfDotProducts < 200) {
    const unsigned int numberOfThreadsPerBlock =
      std::min(unsigned(1024),
               unsigned(ceil(dotProductSize / 32.)) * 32);
    return
      runCudaTest(CudaStyle_Reduction,
                  numberOfThreadsPerBlock,
                  numberOfRepeats,
                  maxNumberOfCudaBlocks,
                  numberOfDotProducts,
                  maxNumberOfDotProducts,
                  dotProductSize,
                  memorySize,
                  correctResults,
                  clearCacheStyle,
                  dev_junkDataToClearTheCache,
                  junkDataSize,
                  dev_dotProductData_LayoutRight_A,
                  dev_dotProductData_LayoutRight_B,
                  dev_junkDataCounter,
                  totalNumberOfRepeats,
                  dev_dotProductResults,
                  dotProductResults);
  } else {
    const unsigned int numberOfThreadsPerBlock = 1024;
    return
      runCudaTest(CudaStyle_Independent,
                  numberOfThreadsPerBlock,
                  numberOfRepeats,
                  maxNumberOfCudaBlocks,
                  numberOfDotProducts,
                  maxNumberOfDotProducts,
                  dotProductSize,
                  memorySize,
                  correctResults,
                  clearCacheStyle,
                  dev_junkDataToClearTheCache,
                  junkDataSize,
                  dev_dotProductData_LayoutLeft_A,
                  dev_dotProductData_LayoutLeft_B,
                  dev_junkDataCounter,
                  totalNumberOfRepeats,
                  dev_dotProductResults,
                  dotProductResults);
  }
}





#ifdef ENABLE_KOKKOS


template <class DeviceType, class KokkosDotProductData>
double
runKokkosTest(const unsigned int numberOfDotProducts,
              const unsigned int numberOfRepeats,
              const unsigned int dotProductSize,
              const unsigned int memorySize,
              const vector<float> & dotProductData_LayoutRight_A,
              const vector<float> & dotProductData_LayoutRight_B,
              const vector<float> & correctResults,
              const string & kokkosFlavor,
              const ClearCacheStyle clearCacheStyle,
              const vector<int> & junkDataToClearTheCache,
              size_t * junkDataCounter,
              unsigned int * const totalNumberOfRepeats,
              vector<float> * dotProductResults) {

  const unsigned int junkDataSize = junkDataToClearTheCache.size();

  typedef typename KokkosDotProductData::HostMirror     KokkosDotProductData_Host;
  typedef Kokkos::View<float*, DeviceType>              KokkosDotProductResults;
  typedef typename KokkosDotProductResults::HostMirror  KokkosDotProductResults_Host;
  typedef Kokkos::View<int*, DeviceType>                KokkosJunkVector;
  typedef typename KokkosJunkVector::HostMirror         KokkosJunkVector_Host;

  KokkosDotProductData dev_kokkosDotProductData_A("kokkos data A",
                                                  numberOfDotProducts,
                                                  dotProductSize);
  KokkosDotProductData_Host kokkosDotProductData_A =
    Kokkos::create_mirror_view(dev_kokkosDotProductData_A);

  KokkosDotProductData dev_kokkosDotProductData_B("kokkos data B",
                                                  numberOfDotProducts,
                                                  dotProductSize);
  KokkosDotProductData_Host kokkosDotProductData_B =
    Kokkos::create_mirror_view(dev_kokkosDotProductData_B);

  KokkosDotProductResults dev_kokkosDotProductResults("kokkos dot product results",
                                                      numberOfDotProducts);
  KokkosDotProductResults_Host kokkosDotProductResults =
    Kokkos::create_mirror_view(dev_kokkosDotProductResults);

  KokkosJunkVector dev_kokkosJunkDataToClearTheCache("kokkos junk data to clear cache",
                                                     junkDataSize);
  KokkosJunkVector_Host kokkosJunkDataToClearTheCache =
    Kokkos::create_mirror_view(dev_kokkosJunkDataToClearTheCache);

  // copy the data into the device views and ship them over
  for (unsigned int dotProductIndex = 0;
       dotProductIndex < numberOfDotProducts; ++dotProductIndex) {
    for (unsigned int entryIndex = 0;
         entryIndex < dotProductSize; ++entryIndex) {
      kokkosDotProductData_A(dotProductIndex, entryIndex) =
        dotProductData_LayoutRight_A[dotProductIndex * dotProductSize +
                                     entryIndex];
      kokkosDotProductData_B(dotProductIndex, entryIndex) =
        dotProductData_LayoutRight_B[dotProductIndex * dotProductSize +
                                     entryIndex];
    }
  }
  Kokkos::deep_copy(dev_kokkosDotProductData_A, kokkosDotProductData_A);
  Kokkos::deep_copy(dev_kokkosDotProductData_B, kokkosDotProductData_B);

  // copy the data into the device views and ship them over
  for (unsigned int junkDataIndex = 0;
       junkDataIndex < junkDataSize; ++junkDataIndex) {
    kokkosJunkDataToClearTheCache(junkDataIndex) =
      junkDataToClearTheCache[junkDataIndex];
  }
  Kokkos::deep_copy(dev_kokkosJunkDataToClearTheCache, kokkosJunkDataToClearTheCache);

  KokkosFunctor_ClearCache<DeviceType,
                           KokkosJunkVector>
    kokkosFunctor_ClearCache(dev_kokkosJunkDataToClearTheCache);

  // breaking formatting convention because holy freak that's long
  KokkosFunctor_Independent<DeviceType,
                            KokkosDotProductData,
                            KokkosDotProductResults>
    kokkosFunctor_Independent(dotProductSize,
                              dev_kokkosDotProductData_A,
                              dev_kokkosDotProductData_B,
                              dev_kokkosDotProductResults);

  timespec tic;
  double totalElapsedTime = 0;
  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
    *totalNumberOfRepeats = *totalNumberOfRepeats + 1;
    if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
         repeatIndex == 1) ||
        clearCacheStyle == ClearCacheAfterEveryRepeat) {
      tic = getTimePoint();
    }

    // actually do the calculation
    Kokkos::parallel_for(numberOfDotProducts, kokkosFunctor_Independent);

    // wait for this repeat's results to finish
    Kokkos::fence();

    if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
      const timespec toc = getTimePoint();
      const float elapsedTime = getElapsedTime(tic, toc);
      totalElapsedTime += elapsedTime;

      // attempt to scrub all levels of cache
      size_t partialJunkDataCounter = 0;
      Kokkos::parallel_reduce(junkDataSize, kokkosFunctor_ClearCache,
                              partialJunkDataCounter);
      *junkDataCounter += partialJunkDataCounter;
    }
  }
  if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
    const timespec toc = getTimePoint();
    totalElapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
  }
  // copy over the results from the device to the host
  Kokkos::deep_copy(kokkosDotProductResults, dev_kokkosDotProductResults);
  for (unsigned int dotProductIndex = 0;
       dotProductIndex < numberOfDotProducts; ++dotProductIndex) {
    dotProductResults->at(dotProductIndex) =
      kokkosDotProductResults(dotProductIndex);
  }
  // check the results
  checkAnswer(correctResults, *dotProductResults,
              dotProductSize, memorySize,
              kokkosFlavor);

  // scrub the results
  std::fill(dotProductResults->begin(),
            dotProductResults->end(),
            std::numeric_limits<float>::quiet_NaN());

  return totalElapsedTime;
}

#endif // ENABLE_KOKKOS



int main(int argc, char* argv[]) {

#ifdef ENABLE_KOKKOS
  Kokkos::initialize(argc, argv);
#endif

  // ===============================================================
  // ********************** < input> ******************************
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
  const vector<unsigned int> dotProductSizes =
    {{8, 16, 32, 64, 128, 256, 512, 1024, 2048}};
  const array<float, 2> memorySizeExtrema = {{1e6, 1e9}};
  const unsigned int numberOfMemorySizes = 20;
  const unsigned int maxNumberOfCudaBlocks = unsigned(1e4);
  const ClearCacheStyle clearCacheStyle =
    ClearCacheAfterEveryRepeat;
  const unsigned int numberOfRepeats =
    (clearCacheStyle == ClearCacheAfterEveryRepeat) ? 10 : 250;
  const string machineName = "shadowfax";
  const string prefix = "data/ArrayOfDotProducts_";
  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  // ********************** </input> ******************************
  // ===============================================================

  // derive some values from the inputs
  const unsigned int numberOfDotProductSizes = dotProductSizes.size();
  const string clearCacheStyleString =
    (clearCacheStyle == ClearCacheAfterEveryRepeat) ? "clearCache" :
    "dontClearCache";
  const string suffix = "_" + clearCacheStyleString + "_" + machineName;

  // create the actual sizes
  vector<unsigned int> memorySizes(numberOfMemorySizes);
  for (unsigned int memorySizeIndex = 0;
       memorySizeIndex < numberOfMemorySizes; ++memorySizeIndex) {
    const float percent = memorySizeIndex / float(numberOfMemorySizes - 1);
    const float minLog = log10(memorySizeExtrema[0]);
    const float maxLog = log10(memorySizeExtrema[1]);
    const float thisLog = minLog + percent * (maxLog - minLog);
    const unsigned int maxDotProductSize = dotProductSizes.back();
    // memory size is linear on a log scale, but rounded to a multiple of the
    //  largest dot product size
    const unsigned int desiredMemorySizeInBytes = pow(10., thisLog);
    // now, in this amount of memory i have to fit two vectors of data
    // that are multiples of the max dot product size
    const unsigned int memorySizeInBytes =
      unsigned(desiredMemorySizeInBytes /
               float(4 * sizeof(float) * maxDotProductSize)) *
      4 * sizeof(float) * maxDotProductSize;
    memorySizes[memorySizeIndex] = memorySizeInBytes;
  }

  // create a c++11 random number generator
  std::mt19937 randomNumberEngine;
  std::uniform_real_distribution<float> randomNumberGenerator(0, 1);

  // these are just containers for storing the numbers we'll be plotting.
  // i feel a little dirty using a vector<vector>, but i don't want to introduce
  //  a dependence on eigen or something for a real matrix.
  vector<vector<float> >
    dotProductSizeMatrix(numberOfDotProductSizes,
                         vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    numberOfDotProductsMatrix(numberOfDotProductSizes,
                              vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    memorySizeMatrix(numberOfDotProductSizes,
                     vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    serialTimesMatrix(numberOfDotProductSizes,
                      vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    ompTimesMatrix(numberOfDotProductSizes,
                   vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    cudaIndependent_TimesMatrix(numberOfDotProductSizes,
                                vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    cudaReduction_TimesMatrix(numberOfDotProductSizes,
                              vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    cudaSwitchingTimesMatrix(numberOfDotProductSizes,
                             vector<float>(numberOfMemorySizes, 0));
#ifdef ENABLE_KOKKOS
  vector<vector<float> >
    kokkosOmpTimesMatrix(numberOfDotProductSizes,
                         vector<float>(numberOfMemorySizes, 0));
  vector<vector<float> >
    kokkosCudaIndependentTimesMatrix(numberOfDotProductSizes,
                                     vector<float>(numberOfMemorySizes, 0));
#endif

  // create some junk data to use in clearing the cache
  size_t junkDataCounter = 0;
  const size_t junkDataSize = 1e7;
  vector<int> junkDataToClearTheCache(junkDataSize, 0);
  for (unsigned int i = 0; i < junkDataSize/100; ++i) {
    junkDataToClearTheCache[(rand() / float(RAND_MAX))*junkDataSize] = 1;
  }
  int * dev_junkDataToClearTheCache;
  checkCudaError(hipMalloc((void **) &dev_junkDataToClearTheCache,
                            junkDataSize * sizeof(int)));
  checkCudaError(hipMemcpy(dev_junkDataToClearTheCache,
                            &junkDataToClearTheCache[0],
                            junkDataSize * sizeof(int),
                            hipMemcpyHostToDevice));
  int * dev_junkDataCounter;
  checkCudaError(hipMalloc((void **) &dev_junkDataCounter,
                            sizeof(int)));
  {
    int temp = 0;
    checkCudaError(hipMemcpy(dev_junkDataCounter,
                              &temp,
                              sizeof(int),
                              hipMemcpyHostToDevice));
  }

  unsigned int totalNumberOfRepeats = 0;

  // for each dot product size
  for (unsigned int dotProductSizeIndex = 0;
       dotProductSizeIndex < numberOfDotProductSizes;
       ++dotProductSizeIndex) {
    const unsigned int dotProductSize = dotProductSizes[dotProductSizeIndex];

    const timespec thisSizesTic = getTimePoint();

    // allocate and initialize the largest amount of memory we'll need, then on
    //  each size we'll just use subsets of this memory.
    const unsigned int maxNumberOfDotProducts =
      memorySizes.back() / 4 / sizeof(float) / dotProductSize;
    vector<float> dotProductData_LayoutRight_A(maxNumberOfDotProducts * dotProductSize);
    vector<float> dotProductData_LayoutRight_B(dotProductData_LayoutRight_A.size());
    vector<float> dotProductData_LayoutLeft_A(dotProductData_LayoutRight_A.size());
    vector<float> dotProductData_LayoutLeft_B(dotProductData_LayoutRight_B.size());
    for (unsigned int dotProductIndex = 0;
         dotProductIndex < maxNumberOfDotProducts; ++dotProductIndex) {
      for (unsigned int entryIndex = 0;
           entryIndex < dotProductSize; ++entryIndex) {
        const unsigned int layoutRightIndex =
          dotProductIndex * dotProductSize + entryIndex;
        dotProductData_LayoutRight_A[layoutRightIndex] =
          randomNumberGenerator(randomNumberEngine);
        dotProductData_LayoutRight_B[layoutRightIndex] =
          randomNumberGenerator(randomNumberEngine);
        const unsigned int layoutLeftIndex =
          entryIndex * maxNumberOfDotProducts + dotProductIndex;
        dotProductData_LayoutLeft_A[layoutLeftIndex] =
          dotProductData_LayoutRight_A[layoutRightIndex];
        dotProductData_LayoutLeft_B[layoutLeftIndex] =
          dotProductData_LayoutRight_B[layoutRightIndex];
      }
    }
    vector<float> dotProductResults(maxNumberOfDotProducts,
                                    std::numeric_limits<float>::quiet_NaN());


    // now, because we'll be working with cuda stuff, also allocate the inputs
    //  and output on the gpu and copy them over
    float * dev_dotProductData_LayoutRight_A;
    checkCudaError(hipMalloc((void **) &dev_dotProductData_LayoutRight_A,
                              maxNumberOfDotProducts * dotProductSize * sizeof(float)));
    checkCudaError(hipMemcpy(dev_dotProductData_LayoutRight_A,
                              &dotProductData_LayoutRight_A[0],
                              maxNumberOfDotProducts * dotProductSize * sizeof(float),
                              hipMemcpyHostToDevice));
    float * dev_dotProductData_LayoutRight_B;
    checkCudaError(hipMalloc((void **) &dev_dotProductData_LayoutRight_B,
                              maxNumberOfDotProducts * dotProductSize * sizeof(float)));
    checkCudaError(hipMemcpy(dev_dotProductData_LayoutRight_B,
                              &dotProductData_LayoutRight_B[0],
                              maxNumberOfDotProducts * dotProductSize * sizeof(float),
                              hipMemcpyHostToDevice));
    float * dev_dotProductResults;
    checkCudaError(hipMalloc((void **) &dev_dotProductResults,
                              maxNumberOfDotProducts * sizeof(float)));
    checkCudaError(hipMemcpy(dev_dotProductResults, &dotProductResults[0],
                              maxNumberOfDotProducts * sizeof(float),
                              hipMemcpyHostToDevice));
    // make and populate the LayoutLeft versions
    float * dev_dotProductData_LayoutLeft_A;
    checkCudaError(hipMalloc((void **) &dev_dotProductData_LayoutLeft_A,
                              maxNumberOfDotProducts * dotProductSize * sizeof(float)));
    checkCudaError(hipMemcpy(dev_dotProductData_LayoutLeft_A,
                              &dotProductData_LayoutLeft_A[0],
                              maxNumberOfDotProducts * dotProductSize * sizeof(float),
                              hipMemcpyHostToDevice));
    float * dev_dotProductData_LayoutLeft_B;
    checkCudaError(hipMalloc((void **) &dev_dotProductData_LayoutLeft_B,
                              maxNumberOfDotProducts * dotProductSize * sizeof(float)));
    checkCudaError(hipMemcpy(dev_dotProductData_LayoutLeft_B,
                              &dotProductData_LayoutLeft_B[0],
                              maxNumberOfDotProducts * dotProductSize * sizeof(float),
                              hipMemcpyHostToDevice));

    // for each memory size
    for (unsigned int memorySizeIndex = 0;
         memorySizeIndex < numberOfMemorySizes;
         ++memorySizeIndex) {
      const unsigned int memorySize = memorySizes[memorySizeIndex];
      const unsigned int numberOfDotProducts =
        memorySize / 4 / sizeof(float) / dotProductSize;
      if (memorySize != 4 * sizeof(float) * numberOfDotProducts * dotProductSize) {
        fprintf(stderr, "invalid memory size of %u for dot product size of "
                "%u because it doesn't divide evenly, remainder is %zu\n",
                memorySize, dotProductSize,
                memorySize % (4 * sizeof(float) * dotProductSize));
        exit(1);
      }

      // ===============================================================
      // ********************** < do serial> ***************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        timespec tic;
        for (unsigned int repeatIndex = 0;
             repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
          ++totalNumberOfRepeats;
          if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
               repeatIndex == 1) ||
              clearCacheStyle == ClearCacheAfterEveryRepeat) {
            tic = getTimePoint();
          }

          // do the actual calculation
          for (unsigned int dotProductIndex = 0;
               dotProductIndex < numberOfDotProducts;
               ++dotProductIndex) {
            const unsigned int shortcutIndex = dotProductIndex * dotProductSize;
            float sum = 0;
            for (unsigned int entryIndex = 0;
                 entryIndex < dotProductSize; ++entryIndex) {
              sum +=
                dotProductData_LayoutRight_A[shortcutIndex + entryIndex] *
                dotProductData_LayoutRight_B[shortcutIndex + entryIndex];
            }
            dotProductResults[dotProductIndex] = sum;
          }

          if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
            const timespec toc = getTimePoint();
            const float elapsedTime = getElapsedTime(tic, toc);
            serialTimesMatrix[dotProductSizeIndex][memorySizeIndex] += elapsedTime;

            junkDataCounter +=
              std::accumulate(junkDataToClearTheCache.begin(),
                              junkDataToClearTheCache.end(), size_t(0));
          }
        }
        if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
          const timespec toc = getTimePoint();
          const float elapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
          serialTimesMatrix[dotProductSizeIndex][memorySizeIndex] = elapsedTime;
        }
      }
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ********************** </do serial> ***************************
      // ===============================================================

      const vector<float> correctResults = dotProductResults;
      // scrub the results
      std::fill(dotProductResults.begin(),
                dotProductResults.end(),
                std::numeric_limits<float>::quiet_NaN());

      // ===============================================================
      // ********************** < do omp> ******************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        timespec tic;
        for (unsigned int repeatIndex = 0;
             repeatIndex < numberOfRepeats + 1; ++repeatIndex) {
          ++totalNumberOfRepeats;
          if ((clearCacheStyle == DontClearCacheAfterEveryRepeat &&
               repeatIndex == 1) ||
              clearCacheStyle == ClearCacheAfterEveryRepeat) {
            tic = getTimePoint();
          }

          // do the actual calculation
#pragma omp parallel for default(none)                                  \
  shared(dotProductData_LayoutRight_A, dotProductData_LayoutRight_B,    \
         dotProductResults)
          for (unsigned int dotProductIndex = 0;
               dotProductIndex < numberOfDotProducts;
               ++dotProductIndex) {
            const unsigned int shortcutIndex = dotProductIndex * dotProductSize;
            float sum = 0;
            for (unsigned int entryIndex = 0;
                 entryIndex < dotProductSize; ++entryIndex) {
              sum +=
                dotProductData_LayoutRight_A[shortcutIndex + entryIndex] *
                dotProductData_LayoutRight_B[shortcutIndex + entryIndex];
            }
            dotProductResults[dotProductIndex] = sum;
          }

          if (clearCacheStyle == ClearCacheAfterEveryRepeat) {
            const timespec toc = getTimePoint();
            const float elapsedTime = getElapsedTime(tic, toc);
            ompTimesMatrix[dotProductSizeIndex][memorySizeIndex] += elapsedTime;

            // attempt to scrub all levels of cache
#pragma omp parallel default(none)                      \
  shared(junkDataCounter, junkDataToClearTheCache)
            {
              const size_t thisThreadsJunkDataCounter =
                std::accumulate(junkDataToClearTheCache.begin(),
                                junkDataToClearTheCache.end(), size_t(0));
              // only one thread adds the junk counter so that the total
              //  at the end is not a function of the number of threads.
#pragma omp single
              junkDataCounter += thisThreadsJunkDataCounter;
            }
          }
        }
        if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
          const timespec toc = getTimePoint();
          const float elapsedTime = getElapsedTime(tic, toc) / numberOfRepeats;
          ompTimesMatrix[dotProductSizeIndex][memorySizeIndex] = elapsedTime;
          // check the results
          checkAnswer(correctResults, dotProductResults,
                      dotProductSize, memorySize,
                      string("omp"));
        }
      }
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ********************** </do omp> ******************************
      // ===============================================================

      // scrub the results
      std::fill(dotProductResults.begin(),
                dotProductResults.end(),
                std::numeric_limits<float>::quiet_NaN());
      checkCudaError(hipMemcpy(dev_dotProductResults, &dotProductResults[0],
                                maxNumberOfDotProducts * sizeof(float),
                                hipMemcpyHostToDevice));

      // ===============================================================
      // ***************** < do cuda independent> **********************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        const unsigned int numberOfThreadsPerBlock = 1024;

        cudaIndependent_TimesMatrix[dotProductSizeIndex][memorySizeIndex] =
          runCudaTest(CudaStyle_Independent,
                      numberOfThreadsPerBlock,
                      numberOfRepeats,
                      maxNumberOfCudaBlocks,
                      numberOfDotProducts,
                      maxNumberOfDotProducts,
                      dotProductSize,
                      memorySize,
                      correctResults,
                      clearCacheStyle,
                      dev_junkDataToClearTheCache,
                      junkDataSize,
                      dev_dotProductData_LayoutLeft_A,
                      dev_dotProductData_LayoutLeft_B,
                      dev_junkDataCounter,
                      &totalNumberOfRepeats,
                      dev_dotProductResults,
                      &dotProductResults);

      }
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ***************** </do cuda independent> **********************
      // ===============================================================

      // ===============================================================
      // ***************** < do cuda reductions> ***********************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
      {
        const unsigned int numberOfThreadsPerBlock =
          std::min(unsigned(1024),
                   unsigned(ceil(dotProductSize / 32.)) * 32);

        cudaReduction_TimesMatrix[dotProductSizeIndex][memorySizeIndex] =
          runCudaTest(CudaStyle_Reduction,
                      numberOfThreadsPerBlock,
                      numberOfRepeats,
                      maxNumberOfCudaBlocks,
                      numberOfDotProducts,
                      maxNumberOfDotProducts,
                      dotProductSize,
                      memorySize,
                      correctResults,
                      clearCacheStyle,
                      dev_junkDataToClearTheCache,
                      junkDataSize,
                      dev_dotProductData_LayoutRight_A,
                      dev_dotProductData_LayoutRight_B,
                      dev_junkDataCounter,
                      &totalNumberOfRepeats,
                      dev_dotProductResults,
                      &dotProductResults);

      }
      cudaSwitchingTimesMatrix[dotProductSizeIndex][memorySizeIndex] =
        runSwitchingCudaTest(numberOfRepeats,
                             maxNumberOfCudaBlocks,
                             numberOfDotProducts,
                             maxNumberOfDotProducts,
                             dotProductSize,
                             memorySize,
                             correctResults,
                             clearCacheStyle,
                             dev_junkDataToClearTheCache,
                             junkDataSize,
                             dev_dotProductData_LayoutLeft_A,
                             dev_dotProductData_LayoutLeft_B,
                             dev_dotProductData_LayoutRight_A,
                             dev_dotProductData_LayoutRight_B,
                             dev_junkDataCounter,
                             &totalNumberOfRepeats,
                             dev_dotProductResults,
                             &dotProductResults);
      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ***************** </do cuda reductions> ***********************
      // ===============================================================

#ifdef ENABLE_KOKKOS
      // ===============================================================
      // ***************** < do kokkos> ********************************
      // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

      {
        typedef Kokkos::OpenMP                             DeviceType;
        typedef Kokkos::View<float**, Kokkos::LayoutRight,
                             DeviceType>                   KokkosDotProductData;
        kokkosOmpTimesMatrix[dotProductSizeIndex][memorySizeIndex] =
          runKokkosTest<DeviceType,
                        KokkosDotProductData>(numberOfDotProducts,
                                              numberOfRepeats,
                                              dotProductSize,
                                              memorySize,
                                              dotProductData_LayoutRight_A,
                                              dotProductData_LayoutRight_B,
                                              correctResults,
                                              string("Kokkos openmp"),
                                              clearCacheStyle,
                                              junkDataToClearTheCache,
                                              &junkDataCounter,
                                              &totalNumberOfRepeats,
                                              &dotProductResults);
      }
      {
        typedef Kokkos::Cuda                               DeviceType;
        typedef Kokkos::View<float**, Kokkos::LayoutLeft,
                             DeviceType>                   KokkosDotProductData;
        // i pass in the layout right version even though this is the cuda
        //  version because it gets copied into the view inside the function.
        kokkosCudaIndependentTimesMatrix[dotProductSizeIndex][memorySizeIndex] =
          runKokkosTest<DeviceType,
                        KokkosDotProductData>(numberOfDotProducts,
                                              numberOfRepeats,
                                              dotProductSize,
                                              memorySize,
                                              dotProductData_LayoutRight_A,
                                              dotProductData_LayoutRight_B,
                                              correctResults,
                                              string("Kokkos cuda"),
                                              clearCacheStyle,
                                              junkDataToClearTheCache,
                                              &junkDataCounter,
                                              &totalNumberOfRepeats,
                                              &dotProductResults);
      }

      // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
      // ***************** </do kokkos> ********************************
      // ===============================================================
#endif // ENABLE_KOKKOS

      dotProductSizeMatrix[dotProductSizeIndex][memorySizeIndex] =
        dotProductSize;
      numberOfDotProductsMatrix[dotProductSizeIndex][memorySizeIndex] =
        numberOfDotProducts;
      memorySizeMatrix[dotProductSizeIndex][memorySizeIndex] =
        memorySize;

    }

    const timespec thisSizesToc = getTimePoint();
    const float thisSizesElapsedTime =
      getElapsedTime(thisSizesTic, thisSizesToc);
    printf("completed %4u repeats of dot products of size %4u "
           "in %7.2f seconds\n", numberOfRepeats,
           dotProductSize, thisSizesElapsedTime);

    checkCudaError(hipFree(dev_dotProductData_LayoutLeft_A));
    checkCudaError(hipFree(dev_dotProductData_LayoutLeft_B));
    checkCudaError(hipFree(dev_dotProductData_LayoutRight_A));
    checkCudaError(hipFree(dev_dotProductData_LayoutRight_B));
    checkCudaError(hipFree(dev_dotProductResults));

  }
  writeTimesMatrixToFile(dotProductSizeMatrix,
                         prefix + string("dotProductSize") + suffix);
  writeTimesMatrixToFile(numberOfDotProductsMatrix,
                         prefix + string("numberOfDotProducts") + suffix);
  writeTimesMatrixToFile(memorySizeMatrix,
                         prefix + string("memorySize") + suffix);
  writeTimesMatrixToFile(serialTimesMatrix,
                         prefix + string("serialTimes") + suffix);
  writeTimesMatrixToFile(ompTimesMatrix,
                         prefix + string("ompTimes") + suffix);
  writeTimesMatrixToFile(cudaIndependent_TimesMatrix,
                         prefix + string("cudaIndependentTimes") + suffix);
  writeTimesMatrixToFile(cudaReduction_TimesMatrix,
                         prefix + string("cudaReductionTimes") + suffix);
  writeTimesMatrixToFile(cudaSwitchingTimesMatrix,
                         prefix + string("cudaSwitchingTimes") + suffix);
#ifdef ENABLE_KOKKOS
  writeTimesMatrixToFile(kokkosOmpTimesMatrix,
                         prefix + string("kokkosOmpTimes") + suffix);
  writeTimesMatrixToFile(kokkosCudaIndependentTimesMatrix,
                         prefix + string("kokkosCudaIndependentTimes") + suffix);
#endif

#ifdef ENABLE_KOKKOS
  const unsigned int numberOfMethods = 7;
#else
  const unsigned int numberOfMethods = 5;
#endif

  const size_t junkDataSum =
    std::accumulate(junkDataToClearTheCache.begin(),
                    junkDataToClearTheCache.end(), size_t(0));
  {
    int temp = 0;
    checkCudaError(hipMemcpy(&temp,
                              dev_junkDataCounter,
                              sizeof(int),
                              hipMemcpyDeviceToHost));
    junkDataCounter += temp;
  }
  if (clearCacheStyle == DontClearCacheAfterEveryRepeat) {
    const size_t expectedDataCounter = 0;
    if (junkDataCounter != expectedDataCounter) {
      fprintf(stderr, "for DontClearCacheAfterEveryRepeat, invalid "
              "junkDataCounter = %zu (%e), it should be %zu (%e)\n",
              junkDataCounter, float(junkDataCounter),
              expectedDataCounter, float(expectedDataCounter));
      exit(1);
    }
  } else {
    const size_t expectedDataCounter =
      junkDataSum * size_t(numberOfMethods) * (numberOfRepeats + 1) * numberOfMemorySizes *
      numberOfDotProductSizes;
    if (junkDataCounter != expectedDataCounter) {
      fprintf(stderr, "for ClearCacheAfterEveryRepeat, invalid "
              "junkDataCounter = %zu (%e), it should be %zu (%e)\n",
              junkDataCounter, float(junkDataCounter),
              expectedDataCounter, float(expectedDataCounter));
      exit(1);
    }
  }

  const unsigned int expectedTotalNumberOfRepeats = numberOfMethods *
    (numberOfRepeats + 1) * numberOfMemorySizes * numberOfDotProductSizes;
  if (totalNumberOfRepeats != expectedTotalNumberOfRepeats) {
    fprintf(stderr, "invalid totalNumberOfRepeats = %u (%e), it should be "
            "%u (%e)\n",
            totalNumberOfRepeats, float(totalNumberOfRepeats),
            expectedTotalNumberOfRepeats, float(expectedTotalNumberOfRepeats));
    exit(1);
  }

#ifdef ENABLE_KOKKOS
  Kokkos::finalize();
#endif

  return 0;
}
